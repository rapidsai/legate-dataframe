#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023-2025, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include <cstdlib>
#include <iostream>
#include <sstream>
#include <stdexcept>
#include <string>

#include <hip/hip_runtime_api.h>

#include <legate/cuda/hip/hip_runtime.h>

#include <cudf/column/column_factories.hpp>
#include <cudf/copying.hpp>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/type_dispatcher.hpp>
#include <rmm/device_uvector.hpp>

#include <legate_dataframe/core/column.hpp>
#include <legate_dataframe/core/null_mask.hpp>
#include <legate_dataframe/core/print.hpp>
#include <legate_dataframe/core/ranges.hpp>
#include <legate_dataframe/utils.hpp>

namespace legate::dataframe {

namespace {

template <typename T>
T* maybe_bind_buffer(legate::PhysicalStore store, std::size_t size)
{
  T* out;
  if (store.is_unbound_store()) {
    out = store.create_output_buffer<T, 1>(legate::Point<1>(size), true).ptr(0);
  } else {
    auto acc = store.write_accessor<T, 1>();
    assert((store.shape<1>().hi[0] - store.shape<1>().lo[0]) == -1 ||
           acc.accessor.is_dense_row_major(store.shape<1>()));
    assert((store.shape<1>().hi[0] + 1 - store.shape<1>().lo[0]) == size);
    out = acc.ptr(store.shape<1>().lo[0]);
  }
  return out;
}

struct move_into_fn {
  template <typename T, std::enable_if_t<cudf::is_rep_layout_compatible<T>()>* = nullptr>
  void operator()(legate::PhysicalArray& array,
                  const cudf::column_view& column,
                  hipStream_t stream,
                  TaskMemoryResource* mr)
  {
    const auto num_rows = column.size();
    if (array.nullable()) {
      bool* null_mask_ptr = maybe_bind_buffer<bool>(array.null_mask(), num_rows);

      if (column.nullable()) {
        null_mask_bits_to_bools(num_rows, null_mask_ptr, column.null_mask(), stream);
      } else {
        LEGATE_CHECK_CUDA(hipMemsetAsync(
          null_mask_ptr, std::numeric_limits<bool>::max(), num_rows * sizeof(bool), stream));
      }
    }

    if (mr != nullptr) {
      auto mem_alloc = mr->release_buffer(column);
      if (mem_alloc.valid() && array.data().is_unbound_store()) {
        array.data().bind_untyped_data(mem_alloc.buffer(), num_rows);
        return;
      }
    }

    T* data_ptr         = maybe_bind_buffer<T>(array.data(), num_rows);
    const T* source_ptr = column.data<T>();
    // hipPointerAttribute_t attr;
    // LEGATE_CHECK_CUDA(hipPointerGetAttributes(&attr, data_ptr));
    // LEGATE_CHECK_CUDA(hipStreamSynchronize(stream));
    LEGATE_CHECK_CUDA(hipMemcpyAsync(
      data_ptr, source_ptr, num_rows * sizeof(T), hipMemcpyDeviceToDevice, stream));
    // LEGATE_CHECK_CUDA(hipDeviceSynchronize());
  }

  template <typename T, std::enable_if_t<std::is_same_v<T, cudf::string_view>>* = nullptr>
  void operator()(legate::PhysicalArray& array,
                  const cudf::column_view& column,
                  hipStream_t stream,
                  TaskMemoryResource* mr)
  {
    // The string version currently doesn't support already bound chars outputs.  Presumably, this
    // can't happen right now anyway, because the result size is not fixed so it should fail early?
    const auto num_rows = column.size();

    if (array.nullable()) {
      bool* null_mask_ptr = maybe_bind_buffer<bool>(array.null_mask(), num_rows);

      if (column.nullable()) {
        null_mask_bits_to_bools(num_rows, null_mask_ptr, column.null_mask(), stream);
      } else {
        LEGATE_CHECK_CUDA(hipMemsetAsync(
          null_mask_ptr, std::numeric_limits<bool>::max(), num_rows * sizeof(bool), stream));
      }
    }

    cudf::strings_column_view str_col{column};
    legate::StringPhysicalArray ary = array.as_string_array();

    if (str_col.size() == 0) {
      if (ary.ranges().data().is_unbound_store()) { ary.ranges().data().bind_empty_data(); }
      if (ary.chars().data().is_unbound_store()) { ary.chars().data().bind_empty_data(); }
      return;
    }

    auto ranges_size = str_col.offsets().size() - 1;
    auto ranges_ptr  = maybe_bind_buffer<legate::Rect<1>>(ary.ranges().data(), ranges_size);
    cudf_offsets_to_local_ranges(ranges_size, ranges_ptr, str_col.offsets(), stream);

    if (str_col.offsets().offset() != 0) {
      throw std::runtime_error("string column seems sliced, which is currently not supported.");
    }
    auto nbytes = str_col.chars_size(stream);
    // NOTE: a string array can never have it's chars data already bound (size may change).
    if (mr != nullptr) {
      // If valid allocation, don't copy the string data.
      auto mem_alloc = mr->release_buffer(str_col, stream);
      if (mem_alloc.valid() && ary.chars().data().is_unbound_store()) {
        ary.chars().data().bind_untyped_data(mem_alloc.buffer(), nbytes);
        return;
      }
    }

    auto chars_ptr = maybe_bind_buffer<int8_t>(ary.chars().data(), nbytes);
    LEGATE_CHECK_CUDA(hipMemcpyAsync(
      chars_ptr, str_col.chars_begin(stream), nbytes, hipMemcpyDeviceToDevice, stream));
  }

  template <typename T,
            std::enable_if_t<!(cudf::is_rep_layout_compatible<T>() ||
                               std::is_same_v<T, cudf::string_view>)>* = nullptr>
  void operator()(legate::PhysicalArray& array,
                  const cudf::column_view& column,
                  hipStream_t stream,
                  TaskMemoryResource* mr)
  {
    // TODO: support lists
    throw std::invalid_argument("move_into(): type not supported");
  }
};

/*
 * Helper to either bind or copy cudf data into PhysicalArray.
 * Context may be `nullptr` when run outside of a trask.
 * This function may take possession of the column data (and bind it to the array).
 */
void from_cudf(legate::PhysicalArray array,
               const cudf::column_view& column,
               hipStream_t stream,
               TaskMemoryResource* mr = nullptr,
               bool scalar            = false)
{
  // Expect the types to match
  if (array.type() != to_legate_type(column.type().id())) {
    throw std::invalid_argument("from_cudf(): type mismatch.");
  }
  // NOTE(seberg): In some cases (replace nulls) we expect no nulls, but
  //     seem to get a nullable column.  So also check `has_nulls()`.
  if (column.nullable() && !array.nullable() && column.has_nulls()) {
    throw std::invalid_argument(
      "from_cudf(): the cudf column is nullable while the PhysicalArray isn't");
  }

  if (scalar && column.size() != 1) {
    throw std::invalid_argument("from_cudf(): scalar column must have size one.");
  }
  cudf::type_dispatcher(column.type(), move_into_fn{}, array, column, stream, mr);
}

legate::LogicalArray from_cudf(const cudf::column_view& col, rmm::cuda_stream_view stream)
{
  auto runtime = legate::Runtime::get_runtime();

  auto cudf_nullable = col.nullable();  // could also count nulls
  if (cudf::type_id::STRING == col.type().id()) {
    cudf::strings_column_view str_col{col};
    auto nbytes = str_col.chars_size(stream);
    auto array  = runtime->create_string_array(
      runtime->create_array({std::uint64_t(col.size())}, legate::rect_type(1), cudf_nullable),
      runtime->create_array({std::uint64_t(nbytes)}, legate::int8()));
    from_cudf(array.get_physical_array(legate::mapping::StoreTarget::FBMEM), col, stream);
    return array;
  }
  if (col.num_children() > 0) {
    throw std::invalid_argument("non-string column with children isn't supported");
  }
  auto array = runtime->create_array({std::uint64_t(col.size())},
                                     to_legate_type(col.type().id()),
                                     cudf_nullable,
                                     false /* scalar */);
  from_cudf(array.get_physical_array(legate::mapping::StoreTarget::FBMEM), col, stream);
  return array;
}

legate::LogicalArray from_cudf(const cudf::scalar& scalar, rmm::cuda_stream_view stream)
{
  // NOTE: this goes via a column-view.  Moving data more directly may be
  // preferable (although libcudf could also grow a way to get a column view).
  auto col = cudf::make_column_from_scalar(scalar, 1, stream);
  return from_cudf(col->view(), stream);
}

struct ArrowToPhysicalArrayVisitor {
  ArrowToPhysicalArrayVisitor(legate::PhysicalArray& array) : array_(array) {}
  legate::PhysicalArray& array_;
  template <typename Type>
  arrow::Status Visit(const arrow::NumericArray<Type>& array)
  {
    using T = typename std::decay_t<decltype(array)>::TypeClass::c_type;
    if (sizeof(T) != array_.type().size()) {
      throw std::invalid_argument(
        "move_into(): the arrow column type size doesn't match the PhysicalArray");
    }
    T* out = maybe_bind_buffer<T>(array_.data(), array.length());
    std::memcpy(out, array.raw_values(), array.length() * sizeof(T));
    return arrow::Status::OK();
  }

  template <typename ArrayType,
            std::enable_if_t<std::is_same_v<ArrayType, arrow::StringArray> ||
                             std::is_same_v<ArrayType, arrow::LargeStringArray>>* = nullptr>
  arrow::Status Visit(const ArrayType& array)
  {
    auto legate_string_array = array_.as_string_array();
    auto ranges_size         = array.length();
    auto ranges =
      maybe_bind_buffer<legate::Rect<1>>(legate_string_array.ranges().data(), ranges_size);
    arrow_offsets_to_local_ranges(array, ranges);
    auto nbytes = array.total_values_length();
    auto chars  = maybe_bind_buffer<int8_t>(legate_string_array.chars().data(), nbytes);
    std::memcpy(chars, array.value_data()->data(), nbytes);
    return arrow::Status::OK();
  }
  arrow::Status Visit(const arrow::BooleanArray& array)
  {
    // Boolean array is bit packed
    auto out = maybe_bind_buffer<bool>(array_.data(), array.length());
    for (std::size_t i = 0; i < array.length(); ++i) {
      out[i] = array.Value(i);
    }
    return arrow::Status::OK();
  }
  arrow::Status Visit(const arrow::Array& array)
  {
    return arrow::Status::NotImplemented("Not implemented for array of type ",
                                         array.type()->ToString());
  }
};

// Copy an arrow array into a physical array
// Binds the legate array if it is unbound
void from_arrow(legate::PhysicalArray array,
                std::shared_ptr<arrow::Array> arrow_array,
                bool scalar = false)
{
  if (array.type() != to_legate_type(*arrow_array->type())) {
    throw std::invalid_argument("from_arrow(): type mismatch: " + array.type().to_string() +
                                " != " + arrow_array->type()->ToString());
  }
  if (!array.nullable() && arrow_array->null_count() > 0) {
    throw std::invalid_argument("from_arrow(): arrow array has nulls but column is not nullable.");
  }
  if (scalar && arrow_array->length() != 1) {
    throw std::invalid_argument("from_arrow(): scalar column must have length 1.");
  }

  if (array.nullable()) {
    bool* null_mask;
    // If the array is a string, its null mask lives in a different place
    if (array.type().code() == legate::Type::Code::STRING) {
      null_mask =
        maybe_bind_buffer<bool>(array.as_string_array().null_mask(), arrow_array->length());
    } else {
      null_mask = maybe_bind_buffer<bool>(array.null_mask(), arrow_array->length());
    }

    for (size_t i = 0; i < arrow_array->length(); ++i) {
      null_mask[i] = arrow_array->IsValid(i);
    }
  }

  // Dispatch arrow::Array types
  ArrowToPhysicalArrayVisitor visitor{array};
  auto status = arrow::VisitArrayInline(*arrow_array, &visitor);
  if (!status.ok()) {
    throw std::invalid_argument("from_arrow(): failed to copy arrow array: " + status.ToString());
  }
}

// Copy an arrow array into a logical array
legate::LogicalArray from_arrow(std::shared_ptr<arrow::Array> arrow_array, bool scalar = false)
{
  // Create an unbound logical array
  auto arrow_has_nulls = arrow_array->null_count() > 0;
  auto runtime         = legate::Runtime::get_runtime();
  if (auto string_array = dynamic_cast<arrow::StringArray*>(arrow_array.get())) {
    auto array = runtime->create_string_array(
      runtime->create_array(
        {std::uint64_t(arrow_array->length())}, legate::rect_type(1), arrow_has_nulls),
      runtime->create_array({std::uint64_t(string_array->total_values_length())}, legate::int8()));
    from_arrow(array.get_physical_array(), arrow_array);
    return array;
  } else if (auto large_string_array = dynamic_cast<arrow::LargeStringArray*>(arrow_array.get())) {
    auto array = runtime->create_string_array(
      runtime->create_array(
        {std::uint64_t(arrow_array->length())}, legate::rect_type(1), arrow_has_nulls),
      runtime->create_array({std::uint64_t(large_string_array->total_values_length())},
                            legate::int8()));
    from_arrow(array.get_physical_array(), arrow_array);
    return array;
  }
  auto array = runtime->create_array({std::uint64_t(arrow_array->length())},
                                     to_legate_type(*arrow_array->type()),
                                     arrow_has_nulls,
                                     false /* scalar */);
  from_arrow(array.get_physical_array(), arrow_array);
  return array;
}

legate::LogicalArray from_arrow(std::shared_ptr<arrow::Scalar> scalar)
{
  auto array = ARROW_RESULT(arrow::MakeArrayFromScalar(*scalar, 1));
  return from_arrow(array);
}
}  // namespace

LogicalColumn::LogicalColumn(cudf::column_view cudf_col, rmm::cuda_stream_view stream)
  : LogicalColumn{from_cudf(cudf_col, stream), cudf_col.type(), /* scalar */ false}
{
}

LogicalColumn::LogicalColumn(const cudf::scalar& cudf_scalar, rmm::cuda_stream_view stream)
  : LogicalColumn{from_cudf(cudf_scalar, stream), cudf_scalar.type(), /* scalar */ true}
{
}

LogicalColumn::LogicalColumn(std::shared_ptr<arrow::Array> arrow_array)
  : LogicalColumn{// This type conversion monstrosity can be improved
                  from_arrow(arrow_array),
                  to_cudf_type(arrow_array->type()),
                  /* scalar */ false}
{
}

LogicalColumn::LogicalColumn(std::shared_ptr<arrow::Scalar> arrow_scalar)
  : LogicalColumn{// This type conversion monstrosity can be improved
                  from_arrow(arrow_scalar),
                  to_cudf_type(arrow_scalar->type),
                  /* scalar */ true}
{
}

namespace {

/**
 * @brief Since Legate's get_physical_array() doesn't support device memory, use this function to
 * copy a physical array to device.
 * TODO: If `get_physical_array()` supports device memory this can be replaced.
 */
[[nodiscard]] rmm::device_buffer copy_physical_array_to_device(const PhysicalArray& physical_array,
                                                               hipStream_t stream)
{
  auto host_ary_nbytes = physical_array.shape<1>().volume() * physical_array.type().size();
  auto ret             = rmm::device_buffer(host_ary_nbytes, stream);
  LEGATE_CHECK_CUDA(hipMemcpyAsync(ret.data(),
                                    read_accessor_as_1d_bytes(physical_array.data()),
                                    host_ary_nbytes,
                                    hipMemcpyHostToDevice,
                                    stream));
  return ret;
}

}  // namespace

std::unique_ptr<cudf::column> LogicalColumn::get_cudf(rmm::cuda_stream_view stream,
                                                      rmm::mr::device_memory_resource* mr) const
{
  // TODO(seberg): This function goes via system memory but could use FBMEM in get_physical_array()
  // One way to achieve this may be to refactor PhysicalColumn::column_view() into a helper
  // and reuse that here. (Once we have a column view, copying it seems reasonable.)
  if (array_->nested()) {
    if (array_->type().code() == legate::Type::Code::STRING) {
      const legate::StringPhysicalArray a = array_->get_physical_array().as_string_array();
      const legate::PhysicalArray chars   = a.chars();
      const auto num_chars                = chars.data().shape<1>().volume();

      // Copy and convert the physical array of ranges to a new cudf column
      std::unique_ptr<cudf::column> cudf_offsets = global_ranges_to_cudf_offsets(
        a.ranges(), num_chars, legate::Memory::Kind::SYSTEM_MEM, stream, mr);

      // Copy the physical array of chars to a new cudf column
      auto chars_buf = copy_physical_array_to_device(chars, stream);
      rmm::device_buffer null_mask{};
      cudf::size_type null_count{0};
      if (a.nullable()) {
        null_mask =
          null_mask_bools_to_bits(a.null_mask(), legate::Memory::Kind::SYSTEM_MEM, stream, mr);
        null_count =
          cudf::null_count(static_cast<const cudf::bitmask_type*>(null_mask.data()), 0, num_rows());
      }
      // Create a new string column from ranges and chars
      return cudf::make_strings_column(num_rows(),
                                       std::move(cudf_offsets),
                                       std::move(chars_buf),
                                       null_count,
                                       std::move(null_mask));
    } else {
      throw std::invalid_argument("nested dtype " + array_->type().to_string() +
                                  " isn't supported");
    }
  }
  rmm::device_buffer null_mask{};
  cudf::size_type null_count{0};
  if (array_->nullable()) {
    legate::PhysicalArray ary = array_->get_physical_array();
    null_mask =
      null_mask_bools_to_bits(ary.null_mask(), legate::Memory::Kind::SYSTEM_MEM, stream, mr);
    null_count =
      cudf::null_count(static_cast<const cudf::bitmask_type*>(null_mask.data()), 0, num_rows());
  }
  return std::make_unique<cudf::column>(
    cudf_type_,
    num_rows(),
    copy_physical_array_to_device(array_->get_physical_array(), stream),
    std::move(null_mask),
    null_count);
}

std::shared_ptr<arrow::Array> LogicalColumn::get_arrow() const
{
  if (unbound()) {
    throw std::runtime_error(
      "Cannot call `.arrow_array()` on a unbound LogicalColumn, please bind it using "
      "`.move_into()`");
  }
  if (array_->nested()) {
    if (array_->type().code() == legate::Type::Code::STRING) {
      const legate::StringPhysicalArray a = array_->get_physical_array().as_string_array();
      const legate::PhysicalArray chars   = a.chars();
      const auto num_chars                = chars.data().shape<1>().volume();

      std::shared_ptr<arrow::Buffer> data =
        ARROW_RESULT(arrow::AllocateBuffer(num_chars * sizeof(int8_t)));
      std::memcpy(data->mutable_data(), read_accessor_as_1d_bytes(chars), num_chars);

      std::shared_ptr<arrow::Buffer> null_bitmask;
      if (a.nullable()) { null_bitmask = null_mask_bools_to_bits(a.null_mask()); }

      auto offsets = global_ranges_to_arrow_offsets(a.ranges().data());

      return std::make_shared<arrow::StringArray>(num_rows(), offsets, data, null_bitmask);

    } else {
      throw std::invalid_argument("nested dtype " + array_->type().to_string() +
                                  " isn't supported");
    }
  } else {
    auto physical_array = array_->get_physical_array();
    auto nbytes         = array_->volume() * array_->type().size();
    std::shared_ptr<arrow::Buffer> data;
    if (this->type().code() == legate::Type::Code::BOOL) {
      // Convert to bit packed
      data = null_mask_bools_to_bits(physical_array.data());
    } else {
      data = ARROW_RESULT(arrow::AllocateBuffer(nbytes * sizeof(int8_t)));
      std::memcpy(data->mutable_data(), read_accessor_as_1d_bytes(physical_array.data()), nbytes);
    }
    std::shared_ptr<arrow::Buffer> null_bitmask;
    if (array_->nullable()) { null_bitmask = null_mask_bools_to_bits(physical_array.null_mask()); }
    auto array_data =
      arrow::ArrayData::Make(to_arrow_type(cudf_type_.id()), num_rows(), {null_bitmask, data});
    return arrow::MakeArray(array_data);
  }
}

std::unique_ptr<cudf::scalar> LogicalColumn::get_cudf_scalar(
  rmm::cuda_stream_view stream, rmm::mr::device_memory_resource* mr) const
{
  // NOTE: We could specialize simple scalars here at least.
  auto col = get_cudf(stream, mr);
  if (col->size() != 1) {
    throw std::invalid_argument("only length 1/scalar columns can be converted to scalar.");
  }
  return std::move(cudf::get_element(col->view(), 0));
}

LogicalColumn LogicalColumn::slice(const legate::Slice& slice) const
{
  return LogicalColumn(array_->slice(0, slice), cudf_type_);
}

std::string LogicalColumn::repr(size_t max_num_items) const
{
  std::stringstream ss;
  ss << "LogicalColumn(";
  if (unbound()) {
    ss << "data=unbound, ";
    if (array_->nullable()) { ss << "null_mask=unbound, "; }
    ss << "dtype=" << array_->type();
  } else {
    legate::PhysicalArray ary = array_->get_physical_array();

    // Notice, `get_physical_array()` returns host memory always
    ss << legate::dataframe::repr(
      ary, max_num_items, legate::Memory::Kind::SYSTEM_MEM, hipStream_t{0});
  }
  if (unbound() || num_rows() == 1) { ss << ", is_scalar=" << (is_scalar() ? "True" : "False"); }
  ss << ")";
  return ss.str();
}

namespace task {

std::string PhysicalColumn::repr(legate::Memory::Kind mem_kind,
                                 hipStream_t stream,
                                 size_t max_num_items) const
{
  std::stringstream ss;
  ss << "PhysicalColumn(";
  ss << legate::dataframe::repr(array_, max_num_items, mem_kind, stream) << ")";
  return ss.str();
}

cudf::column_view PhysicalColumn::column_view() const
{
  if (unbound()) {
    throw std::runtime_error(
      "Cannot call `.column_view()` on a unbound LogicalColumn, please bind it using "
      "`.move_into()`");
  }

  const void* data                    = nullptr;
  const cudf::bitmask_type* null_mask = nullptr;
  cudf::size_type null_count          = 0;
  cudf::size_type offset              = 0;
  std::vector<cudf::column_view> children;

  if (array_.nested()) {
    if (array_.type().code() == legate::Type::Code::STRING) {
      const legate::StringPhysicalArray a = array_.as_string_array();
      const legate::PhysicalArray chars   = a.chars();
      const auto num_chars                = chars.data().shape<1>().volume();

      std::unique_ptr<cudf::column> cudf_offsets = global_ranges_to_cudf_offsets(
        a.ranges(), num_chars, legate::Memory::Kind::GPU_FB_MEM, ctx_->stream(), ctx_->mr());

      // To keep the offsets alive beyond this function, we push it to temporaries before
      // adding it as the first child.
      tmp_cols_.push_back(std::move(cudf_offsets));
      children.push_back(tmp_cols_.back()->view());

      // The second child is the character column
      data = read_accessor_as_1d_bytes(chars.data());
    } else {
      throw std::invalid_argument("nested dtype " + array_.type().to_string() + " isn't supported");
    }
  } else {
    data = read_accessor_as_1d_bytes(array_.data());
  }
  if (array_.nullable()) {
    tmp_null_masks_.push_back(null_mask_bools_to_bits(
      array_.null_mask(), legate::Memory::Kind::GPU_FB_MEM, ctx_->stream(), ctx_->mr()));
    null_mask  = static_cast<const cudf::bitmask_type*>(tmp_null_masks_.back().data());
    null_count = cudf::null_count(null_mask, 0, num_rows(), ctx_->stream());
  }
  return cudf::column_view(cudf_type_, num_rows(), data, null_mask, null_count, offset, children);
}

std::shared_ptr<arrow::Array> PhysicalColumn::arrow_array_view() const
{
  if (unbound()) {
    throw std::runtime_error(
      "Cannot call `.arrow_array()` on a unbound LogicalColumn, please bind it using "
      "`.move_into()`");
  }
  if (array_.nested()) {
    if (array_.type().code() == legate::Type::Code::STRING) {
      const legate::StringPhysicalArray a = array_.as_string_array();
      const legate::PhysicalArray chars   = a.chars();
      auto num_chars                      = chars.data().shape<1>().volume();
      // Its possible to have an empty string, in which we want to avoid giving arrow a null ptr
      std::shared_ptr<arrow::Buffer> data;
      if (num_chars == 0) {
        data = ARROW_RESULT(arrow::AllocateBuffer(1));
      } else {
        data = std::make_shared<arrow::Buffer>(
          reinterpret_cast<const uint8_t*>(read_accessor_as_1d_bytes(chars)), num_chars);
      }

      std::shared_ptr<arrow::Buffer> null_bitmask;
      if (a.nullable()) { null_bitmask = null_mask_bools_to_bits(array_.null_mask()); }

      auto offsets = global_ranges_to_arrow_offsets(a.ranges().data());

      return std::make_shared<arrow::StringArray>(num_rows(), offsets, data, null_bitmask);

    } else {
      throw std::invalid_argument("nested dtype " + array_.type().to_string() + " isn't supported");
    }
  } else {
    auto nbytes = array_.shape<1>().volume() * array_.type().size();
    // 1. Create arrow data buffer - try to use the existing data
    std::shared_ptr<arrow::Buffer> buffer;
    if (this->type().code() == legate::Type::Code::BOOL) {
      // Arrow stores bool bit packed so we must copy
      buffer = null_mask_bools_to_bits(array_.data());
    } else {
      // For other types, we can use the existing data directly
      buffer = std::make_shared<arrow::Buffer>(
        reinterpret_cast<const uint8_t*>(read_accessor_as_1d_bytes(array_.data())), nbytes);
    }

    // 2. Handle null mask
    std::shared_ptr<arrow::Buffer> null_bitmask;
    if (array_.nullable()) { null_bitmask = null_mask_bools_to_bits(array_.null_mask()); }
    // 3. Create ArrayData from buffer
    auto array_data =
      arrow::ArrayData::Make(to_arrow_type(cudf_type_.id()), num_rows(), {null_bitmask, buffer});
    return arrow::MakeArray(array_data);
  }
}

std::unique_ptr<cudf::scalar> PhysicalColumn::cudf_scalar() const
{
  if (num_rows() != 1) {
    throw std::invalid_argument("can only convert length one columns to scalar.");
  }
  return cudf::get_element(column_view(), 0);
}

void PhysicalColumn::copy_into(std::unique_ptr<cudf::column> column)
{
  // String columns seem tricky, so only check their data for being unbound.
  if (unbound()) {
    throw std::invalid_argument("Cannot call `.copy_into()` on an unbound column.");
  }
  from_cudf(array_, column->view(), ctx_->stream(), ctx_->mr(), scalar_out_);
}

void PhysicalColumn::copy_into(std::unique_ptr<cudf::scalar> scalar)
{
  // NOTE: this goes via a column-view.  Moving data more directly may be
  // preferable (although libcudf could also grow a way to get a column view).
  auto col = cudf::make_column_from_scalar(*scalar, 1, ctx_->stream());
  copy_into(std::move(col));
}

void PhysicalColumn::copy_into(std::shared_ptr<arrow::Array> column)
{
  // String columns seem tricky, so only check their data for being unbound.
  if (unbound()) {
    throw std::invalid_argument("Cannot call `.copy_into()` on an unbound column.");
  }
  // TODO: this copies the data, we ideally want to move the arrow buffer.
  from_arrow(array_, column, scalar_out_);
}

void PhysicalColumn::move_into(std::unique_ptr<cudf::column> column)
{
  if (!unbound()) { throw std::invalid_argument("Cannot call `.move_into()` on a bound column."); }
  from_cudf(array_, column->view(), ctx_->stream(), ctx_->mr(), scalar_out_);
}

void PhysicalColumn::move_into(std::unique_ptr<cudf::scalar> scalar)
{
  // NOTE: this goes via a column-view.  Moving data more directly may be
  // preferable (although libcudf could also grow a way to get a column view).

  auto col = cudf::make_column_from_scalar(*scalar, 1, ctx_->stream());
  move_into(std::move(col));
}

void PhysicalColumn::move_into(std::shared_ptr<arrow::Array> column)
{
  if (!unbound()) { throw std::invalid_argument("Cannot call `.move_into()` on a bound column."); }
  // TODO: this copies the data, we ideally want to move the arrow buffer.
  from_arrow(array_, column, scalar_out_);
}

void PhysicalColumn::bind_empty_data() const
{
  if (!unbound()) {
    throw std::invalid_argument("Cannot call `.bind_empty_data()` on a bound column");
  }

  if (scalar_out_) {
    throw std::logic_error("Binding empty data to scalar column should not happen?");
  }

  if (array_.nullable()) { array_.null_mask().bind_empty_data(); }
  if (array_.nested()) {
    legate::StringPhysicalArray ary = array_.as_string_array();
    ary.ranges().data().bind_empty_data();
    ary.chars().data().bind_empty_data();
  } else {
    array_.data().bind_empty_data();
  }
}

}  // namespace task

namespace argument {

legate::Variable add_next_input(legate::AutoTask& task, const LogicalColumn& col, bool broadcast)
{
  add_next_scalar(task, static_cast<std::underlying_type_t<cudf::type_id>>(col.cudf_type().id()));
  auto arr      = col.get_logical_array();
  auto variable = task.add_input(arr);
  if (broadcast) { task.add_constraint(legate::broadcast(variable, {0})); }
  return variable;
}

legate::Variable add_next_output(legate::AutoTask& task, const LogicalColumn& col)
{
  add_next_scalar(task, static_cast<std::underlying_type_t<cudf::type_id>>(col.cudf_type().id()));
  // While we don't care much for reading from a scalar column, pass scalar information
  // for outputs to enforce the result having the right size.
  add_next_scalar(task, col.is_scalar());
  auto variable = task.add_output(col.get_logical_array());
  // Output scalars must be broadcast (for inputs alignment should enforce reasonable things).
  // (If needed, we could enforce that only rank 0 can bind a result instead.)
  if (col.is_scalar()) { task.add_constraint(legate::broadcast(variable, {0})); }
  return variable;
}

}  // namespace argument

}  // namespace legate::dataframe
