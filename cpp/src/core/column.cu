#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023-2025, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include <cstdlib>
#include <iostream>
#include <sstream>
#include <stdexcept>
#include <string>

#include <arrow/c/bridge.h> /* for arrow::ImportArray */

#include <hip/hip_runtime_api.h>

#include <legate/cuda/hip/hip_runtime.h>

#include <cudf/column/column_factories.hpp>
#include <cudf/copying.hpp>
#include <cudf/interop.hpp>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/type_dispatcher.hpp>
#include <rmm/device_uvector.hpp>

#include <legate_dataframe/core/column.hpp>
#include <legate_dataframe/core/null_mask.hpp>
#include <legate_dataframe/core/print.hpp>
#include <legate_dataframe/core/ranges.hpp>
#include <legate_dataframe/utils.hpp>

namespace legate::dataframe {

namespace {

struct move_into_fn {
  template <typename T, std::enable_if_t<cudf::is_rep_layout_compatible<T>()>* = nullptr>
  void operator()(legate::PhysicalArray& array,
                  const cudf::column_view& column,
                  hipStream_t stream,
                  TaskMemoryResource* mr)
  {
    const auto num_rows = column.size();
    if (array.nullable()) {
      bool* null_mask_ptr = maybe_bind_buffer<bool>(array.null_mask(), num_rows);

      if (column.nullable()) {
        null_mask_bits_to_bools(num_rows, null_mask_ptr, column.null_mask(), stream);
      } else {
        LEGATE_CHECK_CUDA(hipMemsetAsync(
          null_mask_ptr, std::numeric_limits<bool>::max(), num_rows * sizeof(bool), stream));
      }
    }

    if (mr != nullptr) {
      auto mem_alloc = mr->release_buffer(column);
      if (mem_alloc.valid() && array.data().is_unbound_store()) {
        array.data().bind_untyped_data(mem_alloc.buffer(), num_rows);
        return;
      }
    }

    T* data_ptr         = maybe_bind_buffer<T>(array.data(), num_rows);
    const T* source_ptr = column.data<T>();
    LEGATE_CHECK_CUDA(hipMemcpyAsync(
      data_ptr, source_ptr, num_rows * sizeof(T), hipMemcpyDeviceToDevice, stream));
  }

  template <typename T, std::enable_if_t<std::is_same_v<T, cudf::string_view>>* = nullptr>
  void operator()(legate::PhysicalArray& array,
                  const cudf::column_view& column,
                  hipStream_t stream,
                  TaskMemoryResource* mr)
  {
    // The string version currently doesn't support already bound chars outputs.  Presumably, this
    // can't happen right now anyway, because the result size is not fixed so it should fail early?
    const auto num_rows = column.size();

    if (array.nullable()) {
      bool* null_mask_ptr = maybe_bind_buffer<bool>(array.null_mask(), num_rows);

      if (column.nullable()) {
        null_mask_bits_to_bools(num_rows, null_mask_ptr, column.null_mask(), stream);
      } else {
        LEGATE_CHECK_CUDA(hipMemsetAsync(
          null_mask_ptr, std::numeric_limits<bool>::max(), num_rows * sizeof(bool), stream));
      }
    }

    cudf::strings_column_view str_col{column};
    legate::StringPhysicalArray ary = array.as_string_array();

    if (str_col.size() == 0) {
      if (ary.ranges().data().is_unbound_store()) { ary.ranges().data().bind_empty_data(); }
      if (ary.chars().data().is_unbound_store()) { ary.chars().data().bind_empty_data(); }
      return;
    }

    auto ranges_size = str_col.offsets().size() - 1;
    auto ranges_ptr  = maybe_bind_buffer<legate::Rect<1>>(ary.ranges().data(), ranges_size);
    cudf_offsets_to_local_ranges(ranges_size, ranges_ptr, str_col.offsets(), stream);

    if (str_col.offsets().offset() != 0) {
      throw std::runtime_error("string column seems sliced, which is currently not supported.");
    }
    auto nbytes = str_col.chars_size(stream);
    // NOTE: a string array can never have it's chars data already bound (size may change).
    if (mr != nullptr) {
      // If valid allocation, don't copy the string data.
      auto mem_alloc = mr->release_buffer(str_col, stream);
      if (mem_alloc.valid() && ary.chars().data().is_unbound_store()) {
        ary.chars().data().bind_untyped_data(mem_alloc.buffer(), nbytes);
        return;
      }
    }

    auto chars_ptr = maybe_bind_buffer<int8_t>(ary.chars().data(), nbytes);
    LEGATE_CHECK_CUDA(hipMemcpyAsync(
      chars_ptr, str_col.chars_begin(stream), nbytes, hipMemcpyDeviceToDevice, stream));
  }

  template <typename T,
            std::enable_if_t<!(cudf::is_rep_layout_compatible<T>() ||
                               std::is_same_v<T, cudf::string_view>)>* = nullptr>
  void operator()(legate::PhysicalArray& array,
                  const cudf::column_view& column,
                  hipStream_t stream,
                  TaskMemoryResource* mr)
  {
    // TODO: support lists
    throw std::invalid_argument("move_into(): type not supported");
  }
};

/*
 * Helper to either bind or copy cudf data into PhysicalArray.
 * Context may be `nullptr` when run outside of a trask.
 * This function may take possession of the column data (and bind it to the array).
 */
void from_cudf(legate::PhysicalArray array,
               const cudf::column_view& column,
               hipStream_t stream,
               TaskMemoryResource* mr = nullptr,
               bool scalar            = false)
{
  // Expect the types to match
  if (array.type() != to_legate_type(column.type().id())) {
    throw std::invalid_argument("from_cudf(): type mismatch.");
  }
  // NOTE(seberg): In some cases (replace nulls) we expect no nulls, but
  //     seem to get a nullable column.  So also check `has_nulls()`.
  if (column.nullable() && !array.nullable() && column.has_nulls()) {
    throw std::invalid_argument(
      "from_cudf(): the cudf column is nullable while the PhysicalArray isn't");
  }

  if (scalar && column.size() != 1) {
    throw std::invalid_argument("from_cudf(): scalar column must have size one.");
  }
  cudf::type_dispatcher(column.type(), move_into_fn{}, array, column, stream, mr);
}

legate::LogicalArray from_cudf(const cudf::column_view& col, rmm::cuda_stream_view stream)
{
  auto runtime = legate::Runtime::get_runtime();

  if (runtime->get_machine().count(legate::mapping::TaskTarget::GPU) == 0) {
    /*
     * NOTE: We can probably remove this eventually, it exists currently
     * mainly because a lot of tests are still written using cudf types and
     * run for CPU only as well.
     * I.e. if we can't copy to GPU, copy to CPU via arrow.
     */
    auto device_array = cudf::to_arrow_host(col, stream);
    auto arrow_type   = to_arrow_type(col.type().id());
    auto arrow_array  = ARROW_RESULT(arrow::ImportArray(&device_array->array, arrow_type));
    return detail::from_arrow(arrow_array);
  }

  auto cudf_nullable = col.nullable();  // could also count nulls
  if (cudf::type_id::STRING == col.type().id()) {
    cudf::strings_column_view str_col{col};
    auto nbytes = str_col.chars_size(stream);
    auto array  = runtime->create_string_array(
      runtime->create_array({std::uint64_t(col.size())}, legate::rect_type(1), cudf_nullable),
      runtime->create_array({std::uint64_t(nbytes)}, legate::int8()));
    from_cudf(array.get_physical_array(legate::mapping::StoreTarget::FBMEM), col, stream);
    return array;
  }
  if (col.num_children() > 0) {
    throw std::invalid_argument("non-string column with children isn't supported");
  }
  auto array = runtime->create_array({std::uint64_t(col.size())},
                                     to_legate_type(col.type().id()),
                                     cudf_nullable,
                                     false /* scalar */);
  from_cudf(array.get_physical_array(legate::mapping::StoreTarget::FBMEM), col, stream);
  return array;
}

legate::LogicalArray from_cudf(const cudf::scalar& scalar, rmm::cuda_stream_view stream)
{
  // NOTE: this goes via a column-view.  Moving data more directly may be
  // preferable (although libcudf could also grow a way to get a column view).
  auto col = cudf::make_column_from_scalar(scalar, 1, stream);
  return from_cudf(col->view(), stream);
}

}  // namespace

LogicalColumn::LogicalColumn(cudf::column_view cudf_col, rmm::cuda_stream_view stream)
  : LogicalColumn{from_cudf(cudf_col, stream), cudf_col.type(), /* scalar */ false}
{
}

LogicalColumn::LogicalColumn(const cudf::scalar& cudf_scalar, rmm::cuda_stream_view stream)
  : LogicalColumn{from_cudf(cudf_scalar, stream), cudf_scalar.type(), /* scalar */ true}
{
}

namespace {

/**
 * @brief Since Legate's get_physical_array() doesn't support device memory, use this function to
 * copy a physical array to device.
 * TODO: If `get_physical_array()` supports device memory this can be replaced.
 */
[[nodiscard]] rmm::device_buffer copy_physical_array_to_device(const PhysicalArray& physical_array,
                                                               hipStream_t stream)
{
  auto host_ary_nbytes = physical_array.shape<1>().volume() * physical_array.type().size();
  auto ret             = rmm::device_buffer(host_ary_nbytes, stream);
  LEGATE_CHECK_CUDA(hipMemcpyAsync(ret.data(),
                                    read_accessor_as_1d_bytes(physical_array.data()),
                                    host_ary_nbytes,
                                    hipMemcpyHostToDevice,
                                    stream));
  return ret;
}

}  // namespace

std::unique_ptr<cudf::column> LogicalColumn::get_cudf(rmm::cuda_stream_view stream,
                                                      rmm::mr::device_memory_resource* mr) const
{
  // TODO(seberg): This function goes via system memory but could use FBMEM in get_physical_array()
  // One way to achieve this may be to refactor PhysicalColumn::column_view() into a helper
  // and reuse that here. (Once we have a column view, copying it seems reasonable.)
  if (array_->nested()) {
    if (array_->type().code() == legate::Type::Code::STRING) {
      const legate::StringPhysicalArray a = array_->get_physical_array().as_string_array();
      const legate::PhysicalArray chars   = a.chars();
      const auto num_chars                = chars.data().shape<1>().volume();

      // Copy and convert the physical array of ranges to a new cudf column
      std::unique_ptr<cudf::column> cudf_offsets = global_ranges_to_cudf_offsets(
        a.ranges(), num_chars, legate::Memory::Kind::SYSTEM_MEM, stream, mr);

      // Copy the physical array of chars to a new cudf column
      auto chars_buf = copy_physical_array_to_device(chars, stream);
      rmm::device_buffer null_mask{};
      cudf::size_type null_count{0};
      if (a.nullable()) {
        null_mask =
          null_mask_bools_to_bits(a.null_mask(), legate::Memory::Kind::SYSTEM_MEM, stream, mr);
        null_count =
          cudf::null_count(static_cast<const cudf::bitmask_type*>(null_mask.data()), 0, num_rows());
      }
      // Create a new string column from ranges and chars
      return cudf::make_strings_column(num_rows(),
                                       std::move(cudf_offsets),
                                       std::move(chars_buf),
                                       null_count,
                                       std::move(null_mask));
    } else {
      throw std::invalid_argument("nested dtype " + array_->type().to_string() +
                                  " isn't supported");
    }
  }
  rmm::device_buffer null_mask{};
  cudf::size_type null_count{0};
  if (array_->nullable()) {
    legate::PhysicalArray ary = array_->get_physical_array();
    null_mask =
      null_mask_bools_to_bits(ary.null_mask(), legate::Memory::Kind::SYSTEM_MEM, stream, mr);
    null_count =
      cudf::null_count(static_cast<const cudf::bitmask_type*>(null_mask.data()), 0, num_rows());
  }
  return std::make_unique<cudf::column>(
    this->cudf_type(),
    num_rows(),
    copy_physical_array_to_device(array_->get_physical_array(), stream),
    std::move(null_mask),
    null_count);
}

std::unique_ptr<cudf::scalar> LogicalColumn::get_cudf_scalar(
  rmm::cuda_stream_view stream, rmm::mr::device_memory_resource* mr) const
{
  // NOTE: We could specialize simple scalars here at least.
  auto col = get_cudf(stream, mr);
  if (col->size() != 1) {
    throw std::invalid_argument("only length 1/scalar columns can be converted to scalar.");
  }
  return std::move(cudf::get_element(col->view(), 0));
}

namespace task {

cudf::column_view PhysicalColumn::column_view() const
{
  if (unbound()) {
    throw std::runtime_error(
      "Cannot call `.column_view()` on a unbound LogicalColumn, please bind it using "
      "`.move_into()`");
  }

  const void* data                    = nullptr;
  const cudf::bitmask_type* null_mask = nullptr;
  cudf::size_type null_count          = 0;
  cudf::size_type offset              = 0;
  std::vector<cudf::column_view> children;

  if (array_.nested()) {
    if (array_.type().code() == legate::Type::Code::STRING) {
      const legate::StringPhysicalArray a = array_.as_string_array();
      const legate::PhysicalArray chars   = a.chars();
      const auto num_chars                = chars.data().shape<1>().volume();

      std::unique_ptr<cudf::column> cudf_offsets = global_ranges_to_cudf_offsets(
        a.ranges(), num_chars, legate::Memory::Kind::GPU_FB_MEM, ctx_->stream(), ctx_->mr());

      // To keep the offsets alive beyond this function, we push it to temporaries before
      // adding it as the first child.
      tmp_cols_.push_back(std::move(cudf_offsets));
      children.push_back(tmp_cols_.back()->view());

      // The second child is the character column
      data = read_accessor_as_1d_bytes(chars.data());
    } else {
      throw std::invalid_argument("nested dtype " + array_.type().to_string() + " isn't supported");
    }
  } else {
    data = read_accessor_as_1d_bytes(array_.data());
  }
  if (array_.nullable()) {
    tmp_null_masks_.push_back(null_mask_bools_to_bits(
      array_.null_mask(), legate::Memory::Kind::GPU_FB_MEM, ctx_->stream(), ctx_->mr()));
    null_mask  = static_cast<const cudf::bitmask_type*>(tmp_null_masks_.back().data());
    null_count = cudf::null_count(null_mask, 0, num_rows(), ctx_->stream());
  }
  return cudf::column_view(
    this->cudf_type(), num_rows(), data, null_mask, null_count, offset, children);
}

std::unique_ptr<cudf::scalar> PhysicalColumn::cudf_scalar() const
{
  if (num_rows() != 1) {
    throw std::invalid_argument("can only convert length one columns to scalar.");
  }
  return cudf::get_element(column_view(), 0);
}

void PhysicalColumn::copy_into(std::unique_ptr<cudf::column> column)
{
  // String columns seem tricky, so only check their data for being unbound.
  if (unbound()) {
    throw std::invalid_argument("Cannot call `.copy_into()` on an unbound column.");
  }
  from_cudf(array_, column->view(), ctx_->stream(), ctx_->mr(), scalar_out_);
}

void PhysicalColumn::copy_into(const cudf::column_view& column)
{
  // String columns seem tricky, so only check their data for being unbound.
  if (unbound()) {
    throw std::invalid_argument("Cannot call `.copy_into()` on an unbound column.");
  }
  from_cudf(array_, column, ctx_->stream(), ctx_->mr(), scalar_out_);
}

void PhysicalColumn::copy_into(std::unique_ptr<cudf::scalar> scalar)
{
  // NOTE: this goes via a column-view.  Moving data more directly may be
  // preferable (although libcudf could also grow a way to get a column view).
  auto col = cudf::make_column_from_scalar(*scalar, 1, ctx_->stream());
  copy_into(std::move(col));
}

void PhysicalColumn::move_into(std::unique_ptr<cudf::column> column)
{
  if (!unbound()) { throw std::invalid_argument("Cannot call `.move_into()` on a bound column."); }
  from_cudf(array_, column->view(), ctx_->stream(), ctx_->mr(), scalar_out_);
}

void PhysicalColumn::move_into(std::unique_ptr<cudf::scalar> scalar)
{
  // NOTE: this goes via a column-view.  Moving data more directly may be
  // preferable (although libcudf could also grow a way to get a column view).

  auto col = cudf::make_column_from_scalar(*scalar, 1, ctx_->stream());
  move_into(std::move(col));
}

}  // namespace task
}  // namespace legate::dataframe
