/*
 * Copyright (c) 2023-2025, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cuda/functional>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/transform.h>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>

#include <cudf/table/table_device_view.cuh>

#include <legate_dataframe/core/task_context.hpp>
#include <legate_dataframe/utils.hpp>

namespace legate::dataframe {

namespace {

template <typename T, typename Enable = void>
struct copy_into_transposed_fn {
  template <typename... Args>
  void operator()(Args&&...)
  {
    throw std::runtime_error("copy_into_transposed(): type not supported");
  }
};

struct copy_into_transposed_impl {
  template <typename T>
  void operator()(GPUTaskContext& ctx,
                  legate::PhysicalArray& array,
                  cudf::table_view tbl,
                  size_t offset,
                  legate::Scalar& null_value)
  {
    copy_into_transposed_fn<T>{}(ctx, array, tbl, offset, null_value);
  }
};

template <typename T>
struct copy_into_transposed_fn<T, std::enable_if_t<cudf::is_rep_layout_compatible<T>()>> {
  void operator()(GPUTaskContext& ctx,
                  legate::PhysicalArray& array,
                  cudf::table_view tbl,
                  size_t offset,
                  legate::Scalar& null_value)
  {
    legate::Rect<2> bounds{{offset, 0}, {offset + tbl.num_rows() - 1, tbl.num_columns() - 1}};
    if (bounds.empty()) { return; }

    auto acc = array.data().write_accessor<T, 2, true>();
    if (!acc.accessor.is_dense_row_major(bounds)) {
      throw std::runtime_error("internal error: copy_into_transpose assume C-order store (data).");
    }

    // Similar to cudf's interleave_columns (we don't want to allocate, so avoid it).
    auto device_input = cudf::table_device_view::create(tbl, ctx.stream());

    auto index_begin = thrust::make_counting_iterator<size_t>(0);
    auto index_end   = thrust::make_counting_iterator<size_t>(bounds.volume());

    if (!array.nullable()) {
      auto get_value_func =
        cuda::proclaim_return_type<T>([input   = *device_input,
                                       divisor = tbl.num_columns(),
                                       val     = null_value.value<T>()] __device__(size_t idx) {
          if (input.column(idx % divisor).is_valid(idx / divisor)) {
            return input.column(idx % divisor).element<T>(idx / divisor);
          } else {
            return val;
          }
        });

      thrust::transform(
        rmm::exec_policy(ctx.stream()), index_begin, index_end, acc.ptr(bounds.lo), get_value_func);
    } else {
      // This assumes that for rep_layout_compatible types `.element<T>(idx)` is OK even for masked
      // values.
      auto get_value_func = cuda::proclaim_return_type<T>(
        [input = *device_input, divisor = tbl.num_columns()] __device__(size_t idx) {
          return input.column(idx % divisor).element<T>(idx / divisor);
        });

      thrust::transform(
        rmm::exec_policy(ctx.stream()), index_begin, index_end, acc.ptr(bounds.lo), get_value_func);

      auto get_isvalid_func = cuda::proclaim_return_type<bool>(
        [input = *device_input, divisor = tbl.num_columns()] __device__(size_t idx) {
          return input.column(idx % divisor).is_valid_nocheck(idx / divisor);
        });

      auto mask_acc = array.null_mask().write_accessor<bool, 2, true>();
      if (!mask_acc.accessor.is_dense_row_major(bounds)) {
        throw std::runtime_error(
          "internal error: copy_into_transpose assume C-order store (mask).");
      }

      thrust::transform(rmm::exec_policy(ctx.stream()),
                        index_begin,
                        index_end,
                        mask_acc.ptr(bounds.lo),
                        get_isvalid_func);
    }
  }
};

}  // namespace

void copy_into_tranposed(GPUTaskContext& ctx,
                         legate::PhysicalArray& array,
                         cudf::table_view tbl,
                         size_t offset,
                         legate::Scalar& null_value)
{
  for (auto&& col : tbl) {
    if (to_cudf_type_id(array.type().code()) != col.type().id()) {
      throw std::runtime_error("internal error: column types changed between files?");
    }
  }

  cudf::type_dispatcher(
    tbl.column(0).type(), copy_into_transposed_impl{}, ctx, array, tbl, offset, null_value);
}

}  // namespace legate::dataframe
