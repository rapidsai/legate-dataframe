#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024-2025, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/detail/aggregation/aggregation.hpp>  // cudf::detail::target_type
#include <cudf/groupby.hpp>
#include <cudf/unary.hpp>
#include <legate_dataframe/core/repartition_by_hash.hpp>
#include <legate_dataframe/groupby_aggregation.hpp>

namespace legate::dataframe::task {

std::unique_ptr<cudf::groupby_aggregation> make_groupby_aggregation(cudf::aggregation::Kind kind)
{
  switch (kind) {
    case cudf::aggregation::Kind::SUM: {
      return cudf::make_sum_aggregation<cudf::groupby_aggregation>();
    }
    case cudf::aggregation::Kind::PRODUCT: {
      return cudf::make_product_aggregation<cudf::groupby_aggregation>();
    }
    case cudf::aggregation::Kind::MIN: {
      return cudf::make_min_aggregation<cudf::groupby_aggregation>();
    }
    case cudf::aggregation::Kind::MAX: {
      return cudf::make_max_aggregation<cudf::groupby_aggregation>();
    }
    case cudf::aggregation::Kind::COUNT_VALID: {
      return cudf::make_count_aggregation<cudf::groupby_aggregation>();
    }
    case cudf::aggregation::Kind::MEAN: {
      return cudf::make_mean_aggregation<cudf::groupby_aggregation>();
    }
    // 0 degrees of freedom instead of default 1 to match Arrow's behavior
    case cudf::aggregation::Kind::VARIANCE: {
      return cudf::make_variance_aggregation<cudf::groupby_aggregation>(0);
    }
    case cudf::aggregation::Kind::STD: {
      return cudf::make_std_aggregation<cudf::groupby_aggregation>(0);
    }
    case cudf::aggregation::Kind::MEDIAN: {
      return cudf::make_median_aggregation<cudf::groupby_aggregation>();
    }
    case cudf::aggregation::Kind::NUNIQUE: {
      return cudf::make_nunique_aggregation<cudf::groupby_aggregation>();
    }
    default: {
      throw std::invalid_argument("Unsupported groupby aggregation");
    }
  }
}

cudf::aggregation::Kind arrow_to_cudf_aggregation(const std::string& agg_name)
{
  std::map<std::string, cudf::aggregation::Kind> agg_map = {
    // Direct mappings
    {"sum", cudf::aggregation::Kind::SUM},
    {"product", cudf::aggregation::Kind::PRODUCT},
    {"min", cudf::aggregation::Kind::MIN},
    {"max", cudf::aggregation::Kind::MAX},
    {"count", cudf::aggregation::Kind::COUNT_VALID},
    {"mean", cudf::aggregation::Kind::MEAN},
    {"variance", cudf::aggregation::Kind::VARIANCE},
    {"stddev", cudf::aggregation::Kind::STD},
    {"approximate_median", cudf::aggregation::Kind::MEDIAN},
    {"count_distinct", cudf::aggregation::Kind::NUNIQUE}};

  //  {"count_all", cudf::aggregation::Kind::COUNT_ALL},
  // "count_all" could be supported but needs some work as it has 0 inputs

  // Don't do these as we don't support nested types at the moment
  // {"list", cudf::aggregation::Kind::COLLECT_LIST},
  // {"tdigest", cudf::aggregation::Kind::TDIGEST}

  // Arrow aggregations with no direct cuDF equivalent:
  // any - cudf has as a reduction aggregation but not groupby aggregation
  // all - cudf has as a reduction aggregation but not groupby aggregation
  // distinct
  // first - could map to NTH_ELEMENT with n=0
  // first_last - no equivalent
  // kurtosis - no equivalent
  // last - could map to NTH_ELEMENT with n=-1
  // min_max - no single equivalent (would need separate MIN/MAX)
  // one - no equivalent
  // pivot_wider - no equivalent
  // skew - no equivalent

  // cuDF aggregations with no direct Arrow equivalent:
  // SUM_OF_SQUARES - no equivalent
  // M2 - no equivalent
  // QUANTILE - no equivalent
  // ARGMAX - no equivalent
  // ARGMIN - no equivalent
  // NTH_ELEMENT - no equivalent
  // ROW_NUMBER - no equivalent
  // EWMA - no equivalent
  // RANK - no equivalent
  // COLLECT_SET - no equivalent
  // LEAD - no equivalent
  // LAG - no equivalent
  // PTX - no equivalent
  // CUDA - no equivalent
  // HOST_UDF - no equivalent
  // MERGE_LISTS - no equivalent
  // MERGE_SETS - no equivalent
  // MERGE_M2 - no equivalent
  // COVARIANCE - no equivalent
  // CORRELATION - no equivalent
  // MERGE_TDIGEST - no equivalent
  // HISTOGRAM - no equivalent
  // MERGE_HISTOGRAM - no equivalent
  // BITWISE_AGG - no equivalent
  if (agg_map.count(agg_name) == 0) {
    throw std::invalid_argument("Unsupported aggregation: " + agg_name);
  }
  return agg_map.at(agg_name);
}

/*static*/ void GroupByAggregationTask::gpu_variant(legate::TaskContext context)
{
  TaskContext ctx{context};
  auto table        = argument::get_next_input<PhysicalTable>(ctx);
  auto output       = argument::get_next_output<PhysicalTable>(ctx);
  auto _key_col_idx = argument::get_next_scalar_vector<size_t>(ctx);
  std::vector<cudf::size_type> key_col_idx(_key_col_idx.begin(), _key_col_idx.end());

  // Get the `column_aggs` task argument
  std::vector<std::tuple<size_t, cudf::aggregation::Kind, size_t>> column_aggs;
  auto column_aggs_size = argument::get_next_scalar<size_t>(ctx);
  for (size_t i = 0; i < column_aggs_size; ++i) {
    auto in_col_idx  = argument::get_next_scalar<size_t>(ctx);
    auto kind        = argument::get_next_scalar<std::string>(ctx);
    auto out_col_idx = argument::get_next_scalar<size_t>(ctx);
    column_aggs.push_back({in_col_idx, arrow_to_cudf_aggregation(kind), out_col_idx});
  }

  // Repartition `table` based on the keys such that each node can do a local groupby.
  auto repartitioned = repartition_by_hash(ctx, table.table_view(), key_col_idx);

  // In order to create the aggregation requests, we walk through `column_aggs` and for
  // each unique input-column-index, we create an aggregation request and append the
  // aggregation-kinds found in `column_aggs`.
  std::vector<cudf::groupby::aggregation_request> requests;
  std::map<size_t, std::pair<size_t, size_t>> out_col_to_request_and_agg_idx;
  {
    std::map<size_t, size_t> in_col_to_request_idx;
    for (const auto& [in_col_idx, kind, out_col_idx] : column_aggs) {
      // If this is the first time we see `in_col_idx`, we create a new `aggregation_request`
      // with `values` set to the column of `in_col_idx` and an empty aggregation vector.
      if (in_col_to_request_idx.find(in_col_idx) == in_col_to_request_idx.end()) {
        in_col_to_request_idx[in_col_idx] = requests.size();
        requests.push_back(cudf::groupby::aggregation_request{
          .values = repartitioned->get_column(in_col_idx), .aggregations = {}});
      }

      // Find the `aggregation_request` that belongs to `in_col_idx`
      size_t request_idx = in_col_to_request_idx.at(in_col_idx);
      auto& request      = requests.at(request_idx);
      // Add the aggregation kind to the request
      request.aggregations.push_back(make_groupby_aggregation(kind));

      // Record in which index in `requests` and `request.aggregations`, the
      // aggregation was added.
      out_col_to_request_and_agg_idx[out_col_idx] = {request_idx, request.aggregations.size() - 1};
    }
  }

  // Do a local groupby
  cudf::groupby::groupby gb_obj(repartitioned->select(key_col_idx), cudf::null_policy::INCLUDE);
  auto [unique_keys, agg_result] = gb_obj.aggregate(requests, ctx.stream(), ctx.mr());

  // Gather the output columns. The key columns goes first.
  auto output_columns = unique_keys->release();

  // Then we add the columns in `agg_result` using the order recorded
  // in `out_col_to_request_and_agg_idx`.
  output_columns.resize(output_columns.size() + out_col_to_request_and_agg_idx.size());
  auto out_types = output.cudf_types();
  for (auto [out_col_idx, request_and_agg_idx] : out_col_to_request_and_agg_idx) {
    auto [request_idx, agg_idx] = request_and_agg_idx;

    output_columns.at(out_col_idx) = std::move(agg_result.at(request_idx).results.at(agg_idx));

    // Cast the cudf output to be consistent with the output, which has output types according to
    // arrow convention
    if (output_columns.at(out_col_idx)->type() != out_types.at(out_col_idx)) {
      output_columns.at(out_col_idx) = cudf::cast(
        output_columns.at(out_col_idx)->view(), out_types.at(out_col_idx), ctx.stream(), ctx.mr());
    }
  }

  output.move_into(std::move(output_columns));
}
}  // namespace legate::dataframe::task
