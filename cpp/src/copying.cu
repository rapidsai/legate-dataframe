/*
 * Copyright (c) 2023-2025, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <legate.h>
#include <legate/cuda/hip/hip_runtime.h>

#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/for_each.h>

#include <cudf/copying.hpp>
#include <cudf/types.hpp>

#include <legate_dataframe/copying.hpp>
#include <legate_dataframe/core/column.hpp>
#include <legate_dataframe/core/library.hpp>
#include <legate_dataframe/core/table.hpp>
#include <legate_dataframe/core/task_argument.hpp>
#include <legate_dataframe/core/task_context.hpp>

namespace legate::dataframe::task {

/*static*/ void CopyIfElseTask::gpu_variant(legate::TaskContext context)
{
  TaskContext ctx{context};
  const auto cond = argument::get_next_input<PhysicalColumn>(ctx);
  const auto lhs  = argument::get_next_input<PhysicalColumn>(ctx);
  const auto rhs  = argument::get_next_input<PhysicalColumn>(ctx);
  auto output     = argument::get_next_output<PhysicalColumn>(ctx);

  if (cond.num_rows() <= 0) {
    output.bind_empty_data();
    return;
  }

  std::unique_ptr<cudf::column> ret;
  /*
   * Use scalars if inputs are to ensure broadcasting works, cond is always a column.
   * This unfortunately requires 4 cases (all 4 overloads provided by libcudf).
   */
  if (lhs.num_rows() == 1 && rhs.num_rows() != 1) {
    auto lhs_scalar = lhs.cudf_scalar();
    ret             = cudf::copy_if_else(
      *lhs_scalar, rhs.column_view(), cond.column_view(), ctx.stream(), ctx.mr());
  } else if (rhs.num_rows() == 1 && lhs.num_rows() != 1) {
    auto rhs_scalar = rhs.cudf_scalar();
    ret             = cudf::copy_if_else(
      lhs.column_view(), *rhs_scalar, cond.column_view(), ctx.stream(), ctx.mr());
  } else if (lhs.num_rows() == 1 && rhs.num_rows() == 1) {
    auto lhs_scalar = lhs.cudf_scalar();
    auto rhs_scalar = rhs.cudf_scalar();
    ret = cudf::copy_if_else(*lhs_scalar, *rhs_scalar, cond.column_view(), ctx.stream(), ctx.mr());
  } else {
    ret = cudf::copy_if_else(
      lhs.column_view(), rhs.column_view(), cond.column_view(), ctx.stream(), ctx.mr());
  }

  if (get_prefer_eager_allocations()) {
    output.copy_into(std::move(ret));
  } else {
    output.move_into(std::move(ret));
  }
}

struct copy_store_fn {
  template <legate::Type::Code CODE>
  void operator()(TaskContext& ctx,
                  const legate::PhysicalStore& input,
                  legate::PhysicalStore& output)
  {
    using value_type = legate::type_of_t<CODE>;
    std::array<size_t, 1> in_strides{};
    std::array<size_t, 1> out_strides{};
    auto in_ptr = input.read_accessor<value_type, 1>().ptr(input.shape<1>(), in_strides.data());
    auto out_ptr =
      output.write_accessor<value_type, 1>().ptr(output.shape<1>(), out_strides.data());
    assert(input.shape<1>().volume() == output.shape<1>().volume());
    assert(input.shape<1>().volume() <= 1 || (in_strides[0] == 1 && out_strides[0] == 1));
    LEGATE_CHECK_CUDA(hipMemcpyAsync(out_ptr,
                                      in_ptr,
                                      input.shape<1>().volume() * sizeof(value_type),
                                      hipMemcpyDeviceToDevice,
                                      ctx.stream()));
  }
};

/*static*/ void CopyTask::gpu_variant(legate::TaskContext context)
{
  TaskContext ctx{context};
  const auto input = ctx.get_next_input_arg();
  auto output      = ctx.get_next_output_arg();

  if (input.shape<1>().volume() <= 0) {
    return;  // Nothing to do, but e.g. pointer getting might fail.
  }

  auto in_store  = input.data();
  auto out_store = output.data();
  legate::type_dispatch(input.type().code(), copy_store_fn{}, ctx, in_store, out_store);
  if (input.nullable()) {
    auto null_mask_in_store  = input.null_mask();
    auto null_mask_out_store = output.null_mask();
    copy_store_fn{}.operator()<legate::Type::Code::BOOL>(
      ctx, null_mask_in_store, null_mask_out_store);
  } else if (output.nullable()) {
    auto out_acc = output.null_mask().write_accessor<bool, 1>();
    LEGATE_CHECK_CUDA(hipMemsetAsync(out_acc.ptr(output.shape<1>()),
                                      true,
                                      output.shape<1>().volume() * sizeof(bool),
                                      ctx.stream()));
  }
}

/*static*/ void CopyOffsetsTask::gpu_variant(legate::TaskContext context)
{
  TaskContext ctx{context};
  const auto input = ctx.get_next_input_arg();
  auto output      = ctx.get_next_output_arg();
  auto offset      = argument::get_next_scalar<int64_t>(ctx);

  if (input.shape<1>().volume() <= 0) {
    return;  // Nothing to do, but e.g. pointer getting might fail.
  }

  auto in_store  = input.data();
  auto out_store = output.data();

  std::array<size_t, 1> in_strides{};
  std::array<size_t, 1> out_strides{};
  auto in_ptr =
    input.data().read_accessor<legate::Rect<1>, 1>().ptr(input.shape<1>(), in_strides.data());
  auto out_ptr =
    output.data().write_accessor<legate::Rect<1>, 1>().ptr(output.shape<1>(), out_strides.data());
  assert(input.shape<1>().volume() == output.shape<1>().volume());
  assert(input.shape<1>().volume() <= 1 || (in_strides[0] == 1 && out_strides[0] == 1));

  thrust::transform(thrust::cuda::par.on(ctx.stream()),
                    in_ptr,
                    in_ptr + input.shape<1>().volume(),
                    out_ptr,
                    [offset] __device__(const legate::Rect<1>& rect) {
                      return legate::Rect<1>{rect.lo[0] + offset, rect.hi[0] + offset};
                    });

  if (input.nullable()) {
    auto null_mask_in_store  = input.null_mask();
    auto null_mask_out_store = output.null_mask();
    copy_store_fn{}.operator()<legate::Type::Code::BOOL>(
      ctx, null_mask_in_store, null_mask_out_store);
  } else if (output.nullable()) {
    auto out_acc = output.null_mask().write_accessor<bool, 1>();
    LEGATE_CHECK_CUDA(hipMemsetAsync(out_acc.ptr(output.shape<1>()),
                                      true,
                                      output.shape<1>().volume() * sizeof(bool),
                                      ctx.stream()));
  }
}

}  // namespace legate::dataframe::task
