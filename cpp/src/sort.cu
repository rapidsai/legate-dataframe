#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2025, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <numeric>
#include <stdexcept>
#include <vector>

#include <cudf/binaryop.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/concatenate.hpp>
#include <cudf/copying.hpp>
#include <cudf/filling.hpp>
#include <cudf/merge.hpp>
#include <cudf/replace.hpp>
#include <cudf/scalar/scalar_factories.hpp>
#include <cudf/search.hpp>
#include <cudf/sorting.hpp>
#include <cudf/table/table.hpp>
#include <legate.h>
#include <legate_dataframe/sort.hpp>

#include <legate_dataframe/core/repartition_by_hash.hpp>
#include <legate_dataframe/join.hpp>
#include <legate_dataframe/utils.hpp>

#define DEBUG_SPLITS 0
#if DEBUG_SPLITS
#include <iostream>
#include <sstream>
#endif

namespace legate::dataframe::task {

namespace {
std::unique_ptr<cudf::column> vector_to_column(const std::vector<std::size_t>& vec,
                                               TaskContext& ctx)
{
  auto ncopy = vec.size();
  rmm::device_uvector<std::size_t> split_ind(ncopy, ctx.stream(), ctx.mr());
  LDF_CUDA_TRY(hipMemcpyAsync(split_ind.data(),
                               vec.data(),
                               ncopy * sizeof(std::size_t),
                               hipMemcpyHostToDevice,
                               ctx.stream()));
  LDF_CUDA_TRY(hipStreamSynchronize(ctx.stream()));

  return std::make_unique<cudf::column>(std::move(split_ind), std::move(rmm::device_buffer()), 0);
}

// Create a cudf column with the specified number of rows
template <typename T>
std::unique_ptr<cudf::column> create_column(cudf::size_type num_rows,
                                            T fill_value,
                                            TaskContext& ctx)
{
  if (num_rows == 0) { return cudf::make_empty_column(cudf::data_type{cudf::type_to_id<T>()}); }
  return cudf::sequence(num_rows,
                        *cudf::make_fixed_width_scalar(fill_value, ctx.stream(), ctx.mr()),
                        *cudf::make_fixed_width_scalar(int32_t{0}, ctx.stream(), ctx.mr()),
                        ctx.stream(),
                        ctx.mr());
}

template <typename T>
std::vector<T> column_to_vector(TaskContext& ctx, const cudf::column_view& col)
{
  std::vector<T> ret(col.size());
  if (col.size() > 0) {
    LDF_CUDA_TRY(hipMemcpyAsync(
      ret.data(), col.data<T>(), col.size() * sizeof(T), hipMemcpyDeviceToHost, ctx.stream()));
  }
  return ret;
}

// Extract split points from a sorted table. Add two metadata columns:
//  - the rank of the split point (which worker it came from)
//  - the local index of the split point
std::unique_ptr<cudf::table> extract_local_splits(TaskContext& ctx,
                                                  const cudf::table_view& sorted_table,
                                                  const std::vector<cudf::size_type>& keys_idx)
{
  auto split_values     = get_split_ind(ctx, sorted_table.num_rows(), ctx.nranks, true);
  auto my_split_ind_col = vector_to_column(split_values, ctx);
  auto nsplits          = my_split_ind_col->size();

  auto my_split_rank_col = create_column<int32_t>(nsplits, ctx.rank, ctx);

  auto my_split_cols_tbl = cudf::gather(sorted_table.select(keys_idx),
                                        my_split_ind_col->view(),
                                        cudf::out_of_bounds_policy::DONT_CHECK,
                                        ctx.stream(),
                                        ctx.mr());
  auto table_columns     = my_split_cols_tbl->release();
  table_columns.push_back(std::move(my_split_rank_col));
  table_columns.push_back(std::move(my_split_ind_col));
  return std::make_unique<cudf::table>(std::move(table_columns));
}

std::unique_ptr<cudf::table> merge_distributed_split_candidates(
  TaskContext& ctx,
  const cudf::table_view& local_splits_and_metadata,
  const std::vector<cudf::size_type>& keys_idx,
  const std::vector<cudf::order>& column_order,
  const std::vector<cudf::null_order>& null_precedence)
{
  std::vector<cudf::table_view> exchange_tables;
  for (int i = 0; i < ctx.nranks; i++) {
    exchange_tables.push_back(local_splits_and_metadata);
  }
  auto [split_candidates_shared, owners_split] = shuffle(ctx, exchange_tables, nullptr);

  if (local_splits_and_metadata.num_rows() == 0) {
    // All nodes need to take part in the shuffle (no data here), but the below
    // cannot search a length 0 table, so return immediately.
    return nullptr;
  }

  std::vector<cudf::order> column_orderx(column_order);
  std::vector<cudf::null_order> null_precedencex(null_precedence);
  column_orderx.insert(column_orderx.end(), {cudf::order::ASCENDING, cudf::order::ASCENDING});
  null_precedencex.insert(null_precedencex.end(),
                          {cudf::null_order::AFTER, cudf::null_order::AFTER});

  // Merge is stable as it includes the rank and index in the keys:
  // keys(x) to pick columns from splits (which include rank and index):
  std::vector<cudf::size_type> all_keysx(keys_idx.size() + 2);
  std::iota(all_keysx.begin(), all_keysx.end(), 0);

  auto split_candidates = cudf::merge(
    split_candidates_shared, all_keysx, column_orderx, null_precedencex, ctx.stream(), ctx.mr());
  owners_split.reset();  // No longer need this
  return std::move(split_candidates);
}

std::unique_ptr<cudf::table> extract_global_splits(TaskContext& ctx,
                                                   const cudf::table_view& global_split_candidates)
{
  auto split_indices =
    get_split_ind(ctx, global_split_candidates.num_rows(), ctx.nranks, /* include_start */ false);
  auto split_value_inds = vector_to_column(split_indices, ctx);
  auto split_values     = cudf::gather(global_split_candidates,
                                   split_value_inds->view(),
                                   cudf::out_of_bounds_policy::DONT_CHECK,
                                   ctx.stream(),
                                   ctx.mr());
  return std::move(split_values);
}

std::vector<cudf::size_type> find_destination_ranks(
  TaskContext& ctx,
  const cudf::table_view& sorted_table,
  const cudf::table_view& global_split_values,
  const std::vector<cudf::size_type>& keys_idx,
  const std::vector<cudf::order>& column_order,
  const std::vector<cudf::null_order>& null_precedence

)
{
  std::vector<cudf::size_type> value_keysx(keys_idx.size() + 1);
  std::iota(value_keysx.begin(), value_keysx.end(), 0);
  auto keys_idxx = keys_idx;
  keys_idxx.push_back(sorted_table.num_columns());

  // Create a column with the same length as sorted table, filled with current rank
  auto rank_column = create_column<int32_t>(sorted_table.num_rows(), ctx.rank, ctx);

  // Create a new table view by appending the rank column to the sorted table
  std::vector<cudf::column_view> table_columns;
  for (int i = 0; i < sorted_table.num_columns(); i++) {
    table_columns.push_back(sorted_table.column(i));
  }
  table_columns.push_back(rank_column->view());
  auto sorted_table_with_rank = cudf::table_view(table_columns);

  auto column_order_with_rank = column_order;
  column_order_with_rank.push_back(cudf::order::ASCENDING);
  auto null_precendence_with_rank = null_precedence;
  null_precendence_with_rank.push_back(cudf::null_order::AFTER);
  auto split_indices = cudf::lower_bound(sorted_table_with_rank.select(keys_idxx),
                                         global_split_values.select(value_keysx),
                                         column_order_with_rank,
                                         null_precendence_with_rank,
                                         ctx.stream(),
                                         ctx.mr());

  /*
   * Copy the split candidates to the host and finalize the local splits.
   * (we may have fewer than nranks split-points here and need to pad later.)
   */
  auto splits_indices_host = column_to_vector<cudf::size_type>(ctx, split_indices->view());
  LDF_CUDA_TRY(hipStreamSynchronize(ctx.stream()));
  // In the obscure case where there is less data than ranks, pad split points.
  for (int i = splits_indices_host.size(); i < ctx.nranks - 1; i++) {
    splits_indices_host.push_back(sorted_table.num_rows());
  }

  return splits_indices_host;
}

/*
 * The practical way to do a distributed sort is to use the initial locally
 * sorted table to estimate good split points to shuffle data to the final node.
 *
 * The rough approach for shuffling the data is the following:
 * 1. Extract `nranks` split candidates from the local table and add their rank
 *    and local index.
 * 2. Exchange all split candidate values and sort them
 * 3. Again extract those candidates that evenly split the whole candidate set.
 *    (we do this on all nodes).
 * 4. Shuffle the data based on the final split candidates.
 *
 * This approach is e.g. the same as in cupynumeric.  We cannot guarantee balanced
 * result chunk sizes, but it should ensure results are within 2x the input chunks.
 * If all chunks are balanced and have the same distribution, the result will be
 * (approximately) balanced again.
 *
 * The trickiest thing to take care of are equal values.  Depending which rank
 * the split point came from (i.e. where it is globally from us), we need to pick
 * the split point inde (if ours) or the first equal value or just after the last
 * depending on whether it came from an earlier or later rank.
 */
std::vector<cudf::size_type> find_splits_for_distribution(
  TaskContext& ctx,
  const cudf::table_view& sorted_table,
  const std::vector<cudf::size_type>& keys_idx,
  const std::vector<cudf::order>& column_order,
  const std::vector<cudf::null_order>& null_precedence)
{
  /*
   * Step 1: Extract local candidates and add rank and index information.
   *
   * We use the start index to find the value representing the range
   * (used as a possible split value), but store the corresponding end of the
   * the last step.
   */
  auto local_splits_and_metadata = extract_local_splits(ctx, sorted_table, keys_idx);

  /*
   * Step 2: Share split candidates among all ranks.
   */
  auto global_split_candidates = merge_distributed_split_candidates(
    ctx, local_splits_and_metadata->view(), keys_idx, column_order, null_precedence);

  if (global_split_candidates == nullptr) {
    // Nothing on this worker, we are done
    return {};
  }

  /*
   * Step 3: Find the best splitting points from all candidates
   */
  auto global_split_values = extract_global_splits(ctx, global_split_candidates->view());

  /*
   * Step 4: Find the actual split points for the local dataset.
   *
   */
  return find_destination_ranks(
    ctx, sorted_table, global_split_values->view(), keys_idx, column_order, null_precedence);
}

static std::unique_ptr<cudf::table> apply_limit(TaskContext& ctx,
                                                std::unique_ptr<cudf::table> tbl,
                                                int64_t limit)
{
  if (limit != INT64_MIN && std::abs(limit) < tbl->num_rows()) {
    cudf::size_type cudf_limit = static_cast<cudf::size_type>(limit);
    cudf::table_view slice;
    if (limit < 0) {
      slice =
        cudf::slice(tbl->view(), {tbl->num_rows() + cudf_limit, tbl->num_rows()}, ctx.stream())[0];
    } else {
      slice = cudf::slice(tbl->view(), {0, cudf_limit}, ctx.stream())[0];
    }
    tbl = std::make_unique<cudf::table>(slice);
  }
  return tbl;
}

}  // namespace

/*static*/ void SortTask::gpu_variant(legate::TaskContext context)
{
  TaskContext ctx{context};

  const auto tbl       = argument::get_next_input<PhysicalTable>(ctx);
  const auto keys_idx_ = argument::get_next_scalar_vector<std::size_t>(ctx);
  std::vector<cudf::size_type> keys_idx(keys_idx_.begin(),
                                        keys_idx_.end());  // Change to cudf size type
  const auto sort_ascending = argument::get_next_scalar_vector<bool>(ctx);
  const auto nulls_at_end   = argument::get_next_scalar<bool>(ctx);
  const auto stable         = argument::get_next_scalar<bool>(ctx);
  const auto limit          = argument::get_next_scalar<int64_t>(ctx);
  auto output               = argument::get_next_output<PhysicalTable>(ctx);

  // Convert ordering parameters to cudf types
  std::vector<cudf::order> column_order;
  std::vector<cudf::null_order> null_precedence;
  for (size_t i = 0; i < keys_idx.size(); i++) {
    column_order.push_back(sort_ascending[i] ? cudf::order::ASCENDING : cudf::order::DESCENDING);
    // Flip the null order if the column is descending
    // This makes the result consistent with arrow
    // Otherwise cudf will put nulls at the start of descending columns with
    // cudf::null_order::AFTER
    if (sort_ascending[i] == false) {
      null_precedence.push_back(nulls_at_end ? cudf::null_order::BEFORE : cudf::null_order::AFTER);
    } else {
      null_precedence.push_back(nulls_at_end ? cudf::null_order::AFTER : cudf::null_order::BEFORE);
    }
  }

  // Create a new locally sorted table (we always need this)
  auto cudf_tbl  = tbl.table_view();
  auto key       = cudf_tbl.select(keys_idx);
  auto sort_func = stable ? cudf::stable_sort_by_key : cudf::sort_by_key;
  auto sorted_table =
    sort_func(cudf_tbl, key, column_order, null_precedence, ctx.stream(), ctx.mr());

  sorted_table = apply_limit(ctx, std::move(sorted_table), limit);

  if (ctx.nranks == 1) {
    output.move_into(sorted_table->release());
    return;
  }

  auto split_indices = find_splits_for_distribution(
    ctx, sorted_table->view(), keys_idx, column_order, null_precedence);

  // If the local table has zero rows we cannot split it for sharing and
  // split_indices will be null.  Exchange the (empty) table instead.
  std::vector<cudf::table_view> partitions;
  if (split_indices.size() > 0) {
    partitions = cudf::split(sorted_table->view(), split_indices, ctx.stream());
  } else {
    assert(sorted_table->num_rows() == 0);
    for (int i = 0; i < ctx.nranks; i++) {
      partitions.push_back(sorted_table->view());
    }
  }
  auto [parts, owners] = shuffle(ctx, partitions, std::move(sorted_table));

  std::unique_ptr<cudf::table> result;
  if (!stable) {
    result = cudf::merge(parts, keys_idx, column_order, null_precedence, ctx.stream(), ctx.mr());
  } else {
    // This is not good, but libcudf has no stable merge:
    // https://github.com/rapidsai/cudf/issues/16010
    // https://github.com/rapidsai/cudf/issues/7379
    result = cudf::concatenate(parts, ctx.stream(), ctx.mr());
    owners.reset();  // we created a copy.
    auto res_view = result->view();
    result        = sort_func(
      res_view, res_view.select(keys_idx), column_order, null_precedence, ctx.stream(), ctx.mr());
  }

#if DEBUG_SPLITS
  std::ostringstream result_size_oss;
  result_size_oss << "Rank/chunk " << ctx.rank << " includes " << result->num_rows() << " rows.\n";
  result_size_oss << "    from individual chunks: ";
  for (auto part : parts) {
    result_size_oss << part.num_rows() << ", ";
  }
  std::cout << result_size_oss.str() << std::endl;
#endif
  output.move_into(std::move(result));
}

}  // namespace legate::dataframe::task
