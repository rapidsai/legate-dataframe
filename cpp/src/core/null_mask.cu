#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023-2025, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <limits>

#include <hip/hip_runtime_api.h>
#include <cudf/null_mask.hpp>
#include <cudf/utilities/bit.hpp>
#include <legate_dataframe/core/null_mask.hpp>
#include <legate_dataframe/utils.hpp>

namespace legate::dataframe {

namespace {

__device__ constexpr auto max_bitmask_type = std::numeric_limits<cudf::bitmask_type>::max();

/**
 * @brief CUDA kernel to convert booleans into bits
 *
 * Each CUDA thread writes a whole bitmask word by first setting all bits to 1
 * and then clear the bits that maps to false values in `bools_acc`
 */
template <typename RangesAcc>
__global__ void bools_to_bitmask(int64_t bools_size,
                                 legate::Point<1> bools_lo,
                                 cudf::bitmask_type* bitmask,
                                 RangesAcc bools_acc)
{
  constexpr auto wordsize = cudf::detail::size_in_bits<cudf::bitmask_type>();
  auto word_id            = blockIdx.x * blockDim.x + threadIdx.x;
  auto start_bit          = word_id * wordsize;
  if (start_bit < bools_size) {
    bitmask[word_id] = max_bitmask_type;
    for (auto i = start_bit; i < start_bit + wordsize && i < bools_size; ++i) {
      if (!bools_acc[bools_lo + i]) { cudf::clear_bit_unsafe(bitmask, i); }
    }
  }
}
}  // namespace

[[nodiscard]] rmm::device_buffer null_mask_bools_to_bits(const legate::PhysicalStore& bools,
                                                         legate::Memory::Kind mem_kind,
                                                         rmm::cuda_stream_view stream,
                                                         rmm::mr::device_memory_resource* mr)
{
  auto bools_acc   = bools.read_accessor<bool, 1>();
  auto bools_shape = bools.shape<1>();
  auto bools_size  = bools_shape.volume();

  rmm::device_buffer bitmask =
    cudf::create_null_mask(bools_size, cudf::mask_state::UNINITIALIZED, stream, mr);

  if (bools_size == 0) { return bitmask; }

  // Launch one CUDA thread per bitmask word.
  const int threads_per_block = 256;
  auto num_blocks =
    (cudf::num_bitmask_words(bools_size) + threads_per_block - 1) / threads_per_block;
  if (is_device_mem(mem_kind)) {
    bools_to_bitmask<<<num_blocks, threads_per_block, 0, stream>>>(
      bools_size, bools_shape.lo, static_cast<cudf::bitmask_type*>(bitmask.data()), bools_acc);
  } else {
    auto tmp_dev_buf      = rmm::device_buffer(bools_size * sizeof(bool), stream, mr);
    auto bools_acc_on_dev = static_cast<bool*>(tmp_dev_buf.data());
    LDF_CUDA_TRY(hipMemcpyAsync(bools_acc_on_dev,
                                 bools_acc.ptr(0),
                                 bools_size * sizeof(bool),
                                 hipMemcpyHostToDevice,
                                 stream));
    bools_to_bitmask<<<num_blocks, threads_per_block, 0, stream>>>(
      bools_size, 0, static_cast<cudf::bitmask_type*>(bitmask.data()), bools_acc_on_dev);

    LDF_CUDA_TRY(hipStreamSynchronize(stream));
  }
  return bitmask;
}

namespace {
/**
 * @brief CUDA kernel to convert bits into booleans
 *
 * Each CUDA thread writes a boolean by reading the corresponding bit
 */
__global__ void bitmask_to_bools(int64_t bools_size, bool* bools, const cudf::bitmask_type* bitmask)
{
  auto tid = static_cast<size_t>(blockIdx.x) * blockDim.x + threadIdx.x;
  if (tid >= bools_size) return;
  bools[tid] = cudf::bit_is_set(bitmask, tid);
}
}  // namespace

void null_mask_bits_to_bools(int64_t bools_size,
                             bool* bools,
                             const cudf::bitmask_type* bitmask,
                             rmm::cuda_stream_view stream)
{
  const int threads_per_block = 256;
  auto num_blocks             = (bools_size + threads_per_block - 1) / threads_per_block;
  bitmask_to_bools<<<num_blocks, threads_per_block, 0, stream>>>(bools_size, bools, bitmask);
}

}  // namespace legate::dataframe
