/*
 * Copyright (c) 2023-2025, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <sstream>
#include <stdexcept>

#include <hip/hip_runtime_api.h>

#include "legate/comm/coll.h"
#include <legate.h>
#include <legate/cuda/hip/hip_runtime.h>
#include <legate_dataframe/core/library.hpp>

#include <cudf/concatenate.hpp>
#include <cudf/contiguous_split.hpp>
#include <cudf/copying.hpp>
#include <cudf/detail/contiguous_split.hpp>  // `cudf::detail::pack`
#include <cudf/partitioning.hpp>

#include <legate_dataframe/core/nccl.hpp>
#include <legate_dataframe/core/repartition_by_hash.hpp>
#include <legate_dataframe/core/task_context.hpp>

namespace legate::dataframe::task {

namespace {

/**
 * @brief Help class for exchanging buffer sizes of the packed cudf columns
 */
class ExchangedSizes {
 private:
  legate::Buffer<std::size_t> _all_sizes;
  TaskContext& _ctx;

 public:
  // We use a temporary stream for the metadata communication. This ways, we avoid
  // synchronizing the main task stream.
  hipStream_t stream;

  /**
   * @brief Exchange (all-to-all) the sizes of the packed cudf columns.
   *
   * When constructed, use `metadata` and `gpu_data` to get the buffer size for a specific rank.
   *
   * @param ctx The context of the calling task
   * @param columns A mapping of tasks to their packed columns. E.g. `columns.at(i)`
   * will be send to the i'th task. NB: all tasks beside itself must have a map thus:
   * `columns.size() == ctx.nranks - 1`.
   */
  ExchangedSizes(TaskContext& ctx, const std::map<int, cudf::packed_columns>& columns) : _ctx(ctx)
  {
    LEGATE_CHECK_CUDA(hipStreamCreate(&stream));
    assert(columns.size() == ctx.nranks - 1);
    // Note: Size of this buffer is taken into account in the mapper:
    _all_sizes =
      legate::create_buffer<std::size_t>(ctx.nranks * ctx.nranks * 2, Memory::Kind::Z_COPY_MEM);

    // Copy the sizes of the metadata and gpu_data of each packed column into _all_sizes at the
    // location corresponding to our rank.
    const int stride = ctx.nranks * 2;
    for (int i = 0; i < ctx.nranks; ++i) {
      if (i == ctx.rank) {
        _all_sizes[ctx.rank * stride + i]              = 0;
        _all_sizes[ctx.rank * stride + ctx.nranks + i] = 0;
      } else {
        _all_sizes[ctx.rank * stride + i]              = columns.at(i).metadata->size();
        _all_sizes[ctx.rank * stride + ctx.nranks + i] = columns.at(i).gpu_data->size();
      }
    }

    // We have to sync here before proceeding as we need the sizes to arrive in order to
    // allocate communication buffers later.
    auto task = ctx.get_legate_context();
    task.concurrent_task_barrier();
    CHECK_NCCL(ncclAllGather(_all_sizes.ptr(ctx.rank * stride),
                             _all_sizes.ptr(0),
                             stride,
                             ncclUint64,
                             task_nccl(ctx),
                             stream));
    LEGATE_CHECK_CUDA(hipStreamSynchronize(stream));
    task.concurrent_task_barrier();
  }

  ~ExchangedSizes() { LEGATE_CHECK_CUDA(hipStreamDestroy(stream)); }

  // TODO: implement a destructor that syncs and calls _all_sizes.destroy(). Currently,
  //       the lifespan of `_all_sizes` is until the legate task finish.

  /**
   * @brief Get the size of the metadata send between two ranks.
   *
   * @param src_rank The rank of the sending task
   * @param dst_rank The rank of the receiving task
   * @return Size of the metadata buffer (in bytes)
   */
  std::size_t metadata(int src_rank, int dst_rank)
  {
    return _all_sizes[src_rank * _ctx.nranks * 2 + dst_rank];
  }
  /**
   * @brief Get the size of the GPU data send between two ranks.
   *
   * @param src_rank The rank of the sending task
   * @param dst_rank The rank of the receiving task
   * @return Size of the GPU device buffer (in bytes)
   */
  std::size_t gpu_data(int src_rank, int dst_rank)
  {
    return _all_sizes[src_rank * _ctx.nranks * 2 + _ctx.nranks + dst_rank];
  }
};

}  // namespace

/**
 * @brief Shuffle (all-to-all exchange) packed cudf partitioned table.
 *
 *
 * @param ctx The context of the calling task
 * @param tbl_partitioned The local table partitioned into multiple tables such
 * that `tbl_partitioned.at(i)` should end up at rank i.
 * @param owning_table Optional table owning the data in `tbl_partitioned`.
 * This table is cleaned up early to reduce the peak memory usage.
 * If passed, `tbl_partitioned` is also cleared (as the content is invalid).
 * @return An std::pair where the first entry contains a vector of table_view
 * with all the chunks (including the local copy). The second entry contains
 * a unique_ptr whose contents owns all parts.
 */
std::pair<std::vector<cudf::table_view>,
          std::unique_ptr<std::pair<std::map<int, rmm::device_buffer>, cudf::table>>>
shuffle(TaskContext& ctx,
        std::vector<cudf::table_view>& tbl_partitioned,
        std::unique_ptr<cudf::table> owning_table)
{
  auto context = ctx.get_legate_context();
  if (tbl_partitioned.size() != ctx.nranks) {
    throw std::runtime_error("internal error: partition split has wrong size.");
  }

  // First we pack the columns into contiguous chunks for transfer/shuffling
  // `columns.at(i)` will be send to the i'th task.
  // N.B. all tasks beside itself have a map so `columns.size() == ctx.nranks - 1`.
  std::map<int, cudf::packed_columns> columns;
  for (int i = 0; static_cast<size_t>(i) < tbl_partitioned.size(); ++i) {
    if (i != ctx.rank) {
      columns[i] = cudf::detail::pack(tbl_partitioned[i], ctx.stream(), ctx.mr());
    }
  }
  // Also copy tbl_partitioned.at(ctx.rank).  This copy is unnecessary but allows
  // clearing the (possibly) much larger owning_table (if passed).
  cudf::table local_table(tbl_partitioned.at(ctx.rank), ctx.stream(), ctx.mr());
  if (owning_table) {
    tbl_partitioned.clear();
    owning_table.reset();
  }

  assert(columns.size() == ctx.nranks - 1);
  ExchangedSizes sizes(ctx, columns);

  // Since we a using NCCL, we need to move the metadata of the packed columns to
  // device memory (NCCL only supports GPU and pinned host memory).
  std::map<int, rmm::device_buffer> packed_metadata;
  for (const auto& [peer, col] : columns) {
    packed_metadata.insert(
      {peer,
       rmm::device_buffer(col.metadata->data(), col.metadata->size(), sizes.stream, ctx.mr())});
  }

  // Let's allocate receive buffers for the packed columns.
  // Receive metadata into pinned host memory.
  // Notice, the lifespan of `legate::Buffer` are until the legate task finish,
  // which is fine since we expect the size of the metadata to be small.
  std::map<int, legate::Buffer<uint8_t>> recv_metadata;
  for (int peer = 0; peer < ctx.nranks; ++peer) {
    std::size_t nbytes = sizes.metadata(peer, ctx.rank);
    if (nbytes > 0) {
      assert(peer != ctx.rank);
      // Note: Size of this buffer is taken into account in the mapper:
      recv_metadata.insert(
        {peer, legate::create_buffer<uint8_t>(nbytes, Memory::Kind::Z_COPY_MEM)});
    }
  }
  // Receive gpu_data into device memory (on main task stream).
  std::map<int, rmm::device_buffer> recv_gpu_data;
  for (int peer = 0; peer < ctx.nranks; ++peer) {
    std::size_t nbytes = sizes.gpu_data(peer, ctx.rank);
    if (nbytes > 0) {
      assert(peer != ctx.rank);
      recv_gpu_data.insert({peer, rmm::device_buffer(nbytes, ctx.stream(), ctx.mr())});
    }
  }

  // Perform all-to-all exchange.
  auto task = ctx.get_legate_context();
  task.concurrent_task_barrier();
  CHECK_NCCL(ncclGroupStart());

  // Exchange metadata using the temporary stream `sizes.stream`.
  for (auto& [peer, buf] : recv_metadata) {
    std::size_t nbytes = sizes.metadata(peer, ctx.rank);
    assert(nbytes > 0);
    CHECK_NCCL(ncclRecv(buf.ptr(0), nbytes, ncclInt8, peer, task_nccl(ctx), sizes.stream));
  }
  for (const auto& [peer, buf] : packed_metadata) {
    assert(buf.size() > 0);
    assert(buf.size() == sizes.metadata(ctx.rank, peer));
    CHECK_NCCL(ncclSend(buf.data(), buf.size(), ncclInt8, peer, task_nccl(ctx), sizes.stream));
  }

  // Exchange gpu_data using the task stream `ctx.stream`.
  for (auto& [peer, buf] : recv_gpu_data) {
    std::size_t nbytes = sizes.gpu_data(peer, ctx.rank);
    assert(nbytes > 0);
    CHECK_NCCL(ncclRecv(buf.data(), nbytes, ncclInt8, peer, task_nccl(ctx), ctx.stream()));
  }
  for (const auto& [peer, col] : columns) {
    if (col.gpu_data->size() == 0) { continue; }
    assert(col.gpu_data->size() == sizes.gpu_data(ctx.rank, peer));
    CHECK_NCCL(ncclSend(
      col.gpu_data->data(), col.gpu_data->size(), ncclInt8, peer, task_nccl(ctx), ctx.stream()));
  }
  CHECK_NCCL(ncclGroupEnd());
  task.concurrent_task_barrier();

  // We sync the temporary stream `sizes.stream`, since the unpacking needs the host-side metadata.
  LEGATE_CHECK_CUDA(hipStreamSynchronize(sizes.stream));

  // Let's unpack and return the packed_columns received from our peers
  // (and our own chunk so that `ret` is ordered for stable sorts)
  std::vector<cudf::table_view> ret;
  for (int peer = 0; peer < ctx.nranks; ++peer) {
    if (peer == ctx.rank) {
      ret.push_back(local_table.view());
      continue;
    }
    uint8_t* gpu_data = nullptr;
    if (recv_gpu_data.count(peer)) {
      gpu_data = static_cast<uint8_t*>(recv_gpu_data.at(peer).data());
    }
    ret.push_back(cudf::unpack(recv_metadata.at(peer).ptr(0), gpu_data));
  }

  using owner_t = std::pair<std::map<int, rmm::device_buffer>, cudf::table>;
  return std::make_pair(
    ret,
    std::make_unique<owner_t>(std::make_pair(std::move(recv_gpu_data), std::move(local_table))));
}

std::unique_ptr<cudf::table> repartition_by_hash(
  TaskContext& ctx,
  const cudf::table_view& table,
  const std::vector<cudf::size_type>& columns_to_hash)
{
  /* The goal is to repartition the table based on the hashing of `columns_to_hash`.
   * Our approach:
   *  1) Each task split their local cudf table into `ctx.nranks` partitions based on the
   *     hashing of `columns_to_hash` and assign each partition to a task.
   *  2) Each task pack (serialize) the partitions not assigned to itself.
   *  4) All tasks shuffle (all-to-all exchange) the partitions. `shuffle` does this by first
   *     packing each partition into a contiguous memory block for the transfer.
   *  5) Each task unpack (deserialize) and concatenate the received columns with the self-assigned
   *     partition.
   *  6) Finally, each task return a new local cudf table that contains the concatenated partitions.
   */

  if (ctx.nranks == 1) {
    // TODO: avoid copy
    return std::make_unique<cudf::table>(table, ctx.stream(), ctx.mr());
  }

  // When used, we need to hold on the partition table as long as tbl_partitioned
  std::unique_ptr<cudf::table> partition_table;
  std::vector<cudf::table_view> tbl_partitioned;
  if (table.num_rows() == 0) {
    tbl_partitioned.reserve(ctx.nranks);
    // cudf seems to have issues with splitting (and maybe hash partitioning) empty tables
    for (int i = 0; i < ctx.nranks; i++) {
      tbl_partitioned.push_back(table);
    }
  } else {
    auto res = cudf::hash_partition(table,
                                    columns_to_hash,
                                    ctx.nranks,
                                    cudf::hash_id::HASH_MURMUR3,
                                    cudf::DEFAULT_HASH_SEED,
                                    ctx.stream(),
                                    ctx.mr());
    partition_table.swap(res.first);

    // Notice, the offset argument for split() and hash_partition() doesn't align. hash_partition()
    // returns the start offset of each partition thus we have to skip the first offset.
    // See: <https://github.com/rapidsai/cudf/issues/4607>.
    auto partition_offsets = std::vector<int>(res.second.begin() + 1, res.second.end());

    tbl_partitioned = cudf::split(*partition_table, partition_offsets, ctx.stream());
  }

  auto [tables, owners] = shuffle(ctx, tbl_partitioned, std::move(partition_table));

  return cudf::concatenate(tables, ctx.stream(), ctx.mr());
}

}  // namespace legate::dataframe::task
