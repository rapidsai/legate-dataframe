#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2025, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/aggregation.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/aggregation/aggregation.hpp>
#include <cudf/groupby.hpp>
#include <cudf/reduction.hpp>
#include <cudf/transform.hpp>
#include <cudf/types.hpp>

#include <legate_dataframe/core/library.hpp>
#include <legate_dataframe/reduction.hpp>

namespace legate::dataframe {
namespace task {

namespace {

std::unique_ptr<cudf::reduce_aggregation> make_cudf_reduce_aggregation(const std::string& agg_kind)
{
  if (agg_kind == "sum") {
    return cudf::make_sum_aggregation<cudf::reduce_aggregation>();
  } else if (agg_kind == "product") {
    return cudf::make_product_aggregation<cudf::reduce_aggregation>();
  } else if (agg_kind == "min") {
    return cudf::make_min_aggregation<cudf::reduce_aggregation>();
  } else if (agg_kind == "max") {
    return cudf::make_max_aggregation<cudf::reduce_aggregation>();
  } else if (agg_kind == "mean") {
    return cudf::make_mean_aggregation<cudf::reduce_aggregation>();
  } else {
    throw std::invalid_argument("Unsupported aggregation kind: " + agg_kind);
  }
}
}  // namespace

void ReduceLocalTask::gpu_variant(legate::TaskContext context)
{
  TaskContext ctx{context};

  const auto input = argument::get_next_input<PhysicalColumn>(ctx);
  auto op          = argument::get_next_scalar<std::string>(ctx);
  auto finalize    = argument::get_next_scalar<bool>(ctx);
  auto initial     = argument::get_next_scalar<bool>(ctx);
  auto output      = argument::get_next_output<PhysicalColumn>(ctx);

  // Fetching initial value column below if used.

  auto col_view = input.column_view();
  std::unique_ptr<const cudf::scalar> scalar_res;
  // TODO: Counting is slightly awkward, it may be best if it was just
  // specially handled (once we have a count-valid function)
  if (op == "count_valid") {
    assert(!initial);
    if (!finalize) {
      auto count = col_view.size() - col_view.null_count();
      scalar_res =
        std::make_unique<cudf::scalar_type_t<int64_t>>(count, true, ctx.stream(), ctx.mr());
    } else {
      auto sum   = cudf::make_sum_aggregation<cudf::reduce_aggregation>();
      auto zero  = cudf::numeric_scalar<int64_t>(0, true, ctx.stream(), ctx.mr());
      scalar_res = cudf::reduce(col_view, *sum, output.cudf_type(), zero, ctx.stream(), ctx.mr());
    }
  } else {
    std::unique_ptr<rmm::device_buffer> new_mask;
    auto agg = make_cudf_reduce_aggregation(op);
    // As of 25.08.dev cudfs min/max don't guarantee consistent null handling.
    // Most code uses `nans_to_nulls` early on to deal with this.  We do it very late (i.e. here)
    // currently (also since pyarrow does deal with it).
    if ((op == "min" || op == "max") && cudf::is_floating_point(output.cudf_type())) {
      auto [new_mask_, new_null_count] = cudf::nans_to_nulls(col_view, ctx.stream(), ctx.mr());
      new_mask                         = std::move(new_mask_);
      assert(col_view.num_children() == 0);
      col_view = cudf::column_view(col_view.type(),
                                   col_view.size(),
                                   col_view.head<void>(),
                                   reinterpret_cast<cudf::bitmask_type*>(new_mask->data()),
                                   new_null_count,
                                   col_view.offset());
    }
    if (initial) {
      auto initial_col    = argument::get_next_input<PhysicalColumn>(ctx);
      auto initial_scalar = initial_col.cudf_scalar();
      scalar_res =
        cudf::reduce(col_view, *agg, output.cudf_type(), *initial_scalar, ctx.stream(), ctx.mr());
    } else {
      scalar_res = cudf::reduce(col_view, *agg, output.cudf_type(), ctx.stream(), ctx.mr());
    }
  }

  // Note: cudf has no helper to go to a column view right now, but we could
  // specialize this in principle.
  output.move_into(cudf::make_column_from_scalar(*scalar_res, 1, ctx.stream(), ctx.mr()));
}

}  // namespace task
}  // namespace legate::dataframe
