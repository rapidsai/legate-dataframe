#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024-2025, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <filesystem>
#include <fstream>
#include <stdexcept>
#include <vector>

#include <cudf/concatenate.hpp>
#include <cudf/io/csv.hpp>
#include <legate.h>
#include <rmm/device_buffer.hpp>

#include <legate_dataframe/core/column.hpp>
#include <legate_dataframe/core/library.hpp>
#include <legate_dataframe/core/table.hpp>
#include <legate_dataframe/core/task_context.hpp>

#include <legate_dataframe/csv.hpp>

namespace legate::dataframe::task {

/*static*/ void CSVWrite::gpu_variant(legate::TaskContext context)
{
  TaskContext ctx{context};
  const std::string dirpath  = argument::get_next_scalar<std::string>(ctx);
  const auto column_names    = argument::get_next_scalar_vector<std::string>(ctx);
  const auto tbl             = argument::get_next_input<PhysicalTable>(ctx);
  const std::string filepath = dirpath + "/part." + std::to_string(ctx.rank) + ".csv";
  const auto delimiter       = static_cast<char>(argument::get_next_scalar<int32_t>(ctx));

  auto dest    = cudf::io::sink_info(filepath);
  auto options = cudf::io::csv_writer_options::builder(dest, tbl.table_view());
  options.names(column_names);
  options.inter_column_delimiter(delimiter);

  cudf::io::write_csv(options, ctx.stream());
}

/* static */ void CSVRead::gpu_variant(legate::TaskContext context)
{
  TaskContext ctx{context};
  const auto file_paths       = argument::get_next_scalar_vector<std::string>(ctx);
  const auto column_names     = argument::get_next_scalar_vector<std::string>(ctx);
  const auto use_cols_indexes = argument::get_next_scalar_vector<int>(ctx);
  const auto na_filter        = argument::get_next_scalar<bool>(ctx);
  const auto delimiter        = static_cast<char>(argument::get_next_scalar<int32_t>(ctx));
  const auto nbytes           = argument::get_next_scalar_vector<size_t>(ctx);
  const auto nbytes_total     = argument::get_next_scalar<size_t>(ctx);
  const auto read_header      = argument::get_next_scalar<bool>(ctx);
  PhysicalTable tbl_arg       = argument::get_next_output<PhysicalTable>(ctx);
  argument::get_parallel_launch_task(ctx);

  if (file_paths.size() != nbytes.size()) {
    throw std::runtime_error("internal error: file path and nbytes size mismatch");
  }

  auto [my_bytes_offset, my_num_bytes] = evenly_partition_work(nbytes_total, ctx.rank, ctx.nranks);

  auto dtypes = tbl_arg.cudf_types();

  std::map<std::string, cudf::data_type> dtypes_map;
  for (size_t i = 0; i < dtypes.size(); i++) {
    dtypes_map[column_names[i]] = dtypes[i];
  }

  // Iterate through the file and nrow list and read as many rows from the
  // files as this rank should read while skipping those of the other tasks.
  std::vector<std::unique_ptr<cudf::table>> tables;
  size_t total_bytes_seen = 0;
  for (size_t i = 0; i < file_paths.size() && my_num_bytes > 0; i++) {
    auto file_bytes = nbytes[i];

    if (total_bytes_seen + file_bytes <= my_bytes_offset) {
      // All of this files bytes belong to earlier ranks.
      total_bytes_seen += file_bytes;
      continue;
    }
    // Calculate offset and bytes to read from this file.
    auto file_bytes_offset  = my_bytes_offset - total_bytes_seen;
    auto file_bytes_to_read = std::min(file_bytes - file_bytes_offset, my_num_bytes);

    auto src = cudf::io::source_info(file_paths[i]);
    auto opt = cudf::io::csv_reader_options::builder(src);
    if (file_bytes_offset != 0 || !read_header) {
      // Reading the header makes only sense at the start of a file
      // TODO: If the header is read, could sanity check columns for multiple files.
      opt.header(-1);
    }
    opt.delimiter(delimiter);
    opt.na_filter(na_filter);
    opt.dtypes(dtypes_map);
    opt.byte_range_offset(file_bytes_offset);
    opt.byte_range_size(file_bytes_to_read);
    opt.use_cols_indexes(use_cols_indexes);
    opt.names(column_names);

    auto read_table = cudf::io::read_csv(opt, ctx.stream(), ctx.mr()).tbl;

    // Only add if we read something (otherwise number of cols may be off)
    if (read_table->num_rows() != 0) { tables.emplace_back(std::move(read_table)); }

    // Reading may read additional bytes at the end and less at the start
    // However, there is no need to worry about the actual bytes read,
    // we only worry how much we try to read from the next file.
    my_num_bytes -= file_bytes_to_read;
    my_bytes_offset += file_bytes_to_read;
    total_bytes_seen += file_bytes;
  }

  // Concatenate tables and move the result to the output table
  if (tables.size() == 0) {
    tbl_arg.bind_empty_data();
  } else if (tables.size() == 1) {
    tbl_arg.move_into(std::move(tables.back()));
  } else {
    std::vector<cudf::table_view> table_views;
    for (const auto& table : tables) {
      table_views.push_back(table->view());
    }
    tbl_arg.move_into(cudf::concatenate(table_views, ctx.stream(), ctx.mr()));
  }
}

}  // namespace legate::dataframe::task
