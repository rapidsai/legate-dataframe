#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023-2025, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/types.hpp>
#include <cudf/unary.hpp>
#include <legate.h>

#include <legate_dataframe/core/table.hpp>
#include <legate_dataframe/core/task_argument.hpp>
#include <legate_dataframe/core/task_context.hpp>
#include <legate_dataframe/unaryop.hpp>

namespace legate::dataframe::task {

/*static*/ void CastTask::gpu_variant(legate::TaskContext context)
{
  TaskContext ctx{context};

  const auto input                  = argument::get_next_input<PhysicalColumn>(ctx);
  auto output                       = argument::get_next_output<PhysicalColumn>(ctx);
  cudf::column_view col             = input.column_view();
  std::unique_ptr<cudf::column> ret = cudf::cast(col, output.cudf_type(), ctx.stream(), ctx.mr());
  if (get_prefer_eager_allocations()) {
    output.copy_into(std::move(ret));
  } else {
    output.move_into(std::move(ret));
  }
}

cudf::unary_operator arrow_to_cudf_unary_op(std::string op)
{
  // Arrow unary operators taken from the below list,
  // where an equivalent cudf unary operator exists.
  // https://arrow.apache.org/docs/cpp/compute.html#element-wise-scalar-functions
  // https://docs.rapids.ai/api/libcudf/stable/group__transformation__unaryops
  std::unordered_map<std::string, cudf::unary_operator> arrow_to_cudf_ops = {
    {"sin", cudf::unary_operator::SIN},       {"cos", cudf::unary_operator::COS},
    {"tan", cudf::unary_operator::TAN},       {"asin", cudf::unary_operator::ARCSIN},
    {"acos", cudf::unary_operator::ARCCOS},   {"atan", cudf::unary_operator::ARCTAN},
    {"sinh", cudf::unary_operator::SINH},     {"cosh", cudf::unary_operator::COSH},
    {"tanh", cudf::unary_operator::TANH},     {"asinh", cudf::unary_operator::ARCSINH},
    {"acosh", cudf::unary_operator::ARCCOSH}, {"atanh", cudf::unary_operator::ARCTANH},
    {"exp", cudf::unary_operator::EXP},       {"ln", cudf::unary_operator::LOG},
    {"sqrt", cudf::unary_operator::SQRT},     {"ceil", cudf::unary_operator::CEIL},
    {"floor", cudf::unary_operator::FLOOR},   {"abs", cudf::unary_operator::ABS},
    {"round", cudf::unary_operator::RINT},    {"bit_wise_not", cudf::unary_operator::BIT_INVERT},
    {"invert", cudf::unary_operator::NOT},    {"negate", cudf::unary_operator::NEGATE}};

  if (arrow_to_cudf_ops.find(op) != arrow_to_cudf_ops.end()) { return arrow_to_cudf_ops[op]; }
  throw std::invalid_argument("Could not find cudf binary operator matching: " + op);
  return cudf::unary_operator::ABS;
}

/*static*/ void UnaryOpTask::gpu_variant(legate::TaskContext context)
{
  TaskContext ctx{context};

  auto op               = argument::get_next_scalar<std::string>(ctx);
  const auto input      = argument::get_next_input<PhysicalColumn>(ctx);
  auto output           = argument::get_next_output<PhysicalColumn>(ctx);
  cudf::column_view col = input.column_view();
  std::unique_ptr<cudf::column> ret =
    cudf::unary_operation(col, arrow_to_cudf_unary_op(op), ctx.stream(), ctx.mr());
  if (get_prefer_eager_allocations()) {
    output.copy_into(std::move(ret));
  } else {
    output.move_into(std::move(ret));
  }
}

}  // namespace legate::dataframe::task
