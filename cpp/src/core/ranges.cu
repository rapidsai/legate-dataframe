#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023-2025, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hip/hip_runtime_api.h>
#include <limits>

#include <cudf/column/column_factories.hpp>

#include <legate_dataframe/core/ranges.hpp>
#include <legate_dataframe/utils.hpp>

namespace legate::dataframe {

namespace {

/**
 * @brief CUDA kernel to convert ranges (legate) to offsets (cudf)
 */
template <typename RangesAcc, typename OffsetsAcc>
__global__ void ranges_to_offsets(int64_t offsets_size,
                                  int64_t vardata_size,
                                  legate::Point<1> ranges_shape_lo,
                                  RangesAcc ranges_acc,
                                  OffsetsAcc offsets_acc)
{
  auto tid = static_cast<size_t>(blockIdx.x) * blockDim.x + threadIdx.x;
  if (tid == offsets_size - 1) {
    offsets_acc[tid] = vardata_size;
  } else if (tid < offsets_size) {
    auto global_range_offset = ranges_acc[ranges_shape_lo].lo[0];
    offsets_acc[tid]         = ranges_acc[tid + ranges_shape_lo].lo[0] - global_range_offset;
  }
}

template <typename OffsetsAcc>
std::unique_ptr<cudf::column> global_ranges_to_cudf_offsets_impl(
  const legate::PhysicalArray ranges,
  int64_t num_chars,
  legate::Memory::Kind mem_kind,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr)
{
  using RangeDType  = legate::Rect<1>;
  auto ranges_shape = ranges.data().shape<1>();
  auto ranges_size  = ranges_shape.volume();

  std::unique_ptr<cudf::column> cudf_offsets =
    cudf::make_numeric_column(cudf::data_type{cudf::type_to_id<OffsetsAcc>()},
                              ranges_size + 1,
                              cudf::mask_state::UNALLOCATED,
                              stream,
                              mr);
  OffsetsAcc* offsets_acc     = cudf_offsets->mutable_view().data<OffsetsAcc>();
  const int threads_per_block = 256;
  auto num_blocks             = (cudf_offsets->size() + threads_per_block - 1) / threads_per_block;
  auto ranges_acc             = ranges.data().read_accessor<RangeDType, 1>();

  if (is_device_mem(mem_kind)) {
    ranges_to_offsets<<<num_blocks, threads_per_block, 0, stream>>>(
      cudf_offsets->size(), num_chars, ranges_shape.lo, ranges_acc, offsets_acc);
  } else {
    auto tmp_dev_buf       = rmm::device_buffer(ranges_size * sizeof(RangeDType), stream, mr);
    auto ranges_acc_on_dev = static_cast<RangeDType*>(tmp_dev_buf.data());
    LDF_CUDA_TRY(hipMemcpyAsync(ranges_acc_on_dev,
                                 ranges_acc.ptr(0),
                                 ranges_size * sizeof(RangeDType),
                                 hipMemcpyHostToDevice,
                                 stream));
    ranges_to_offsets<<<num_blocks, threads_per_block, 0, stream>>>(
      cudf_offsets->size(), num_chars, 0, ranges_acc_on_dev, offsets_acc);
    LDF_CUDA_TRY(hipStreamSynchronize(stream));
  }
  return cudf_offsets;
}

}  // namespace

std::unique_ptr<cudf::column> global_ranges_to_cudf_offsets(const legate::PhysicalArray ranges,
                                                            int64_t num_chars,
                                                            legate::Memory::Kind mem_kind,
                                                            rmm::cuda_stream_view stream,
                                                            rmm::mr::device_memory_resource* mr)
{
  if (std::numeric_limits<int32_t>::max() >= num_chars) {
    return global_ranges_to_cudf_offsets_impl<int32_t>(ranges, num_chars, mem_kind, stream, mr);
  } else {
    return global_ranges_to_cudf_offsets_impl<int64_t>(ranges, num_chars, mem_kind, stream, mr);
  }
}

namespace {
/**
 * @brief CUDA kernel to convert offsets (cudf) to ranges (legate)
 */
template <typename OffsetsAcc>
__global__ void offsets_to_ranges(int64_t ranges_size,
                                  legate::Rect<1>* ranges_acc,
                                  const OffsetsAcc* offsets_acc)
{
  auto tid = static_cast<size_t>(blockIdx.x) * blockDim.x + threadIdx.x;
  if (tid >= ranges_size) return;
  auto& range = ranges_acc[tid];
  range.lo[0] = offsets_acc[tid];
  range.hi[0] = offsets_acc[tid + 1] - 1;
}

}  // namespace

void cudf_offsets_to_local_ranges(int64_t ranges_size,
                                  legate::Rect<1>* ranges_acc,
                                  cudf::column_view offsets,
                                  rmm::cuda_stream_view stream)
{
  const int threads_per_block = 256;
  auto num_blocks             = (ranges_size + threads_per_block - 1) / threads_per_block;

  if (offsets.type().id() == cudf::type_id::INT32) {
    offsets_to_ranges<<<num_blocks, threads_per_block, 0, stream>>>(
      ranges_size, ranges_acc, offsets.data<int32_t>());
  } else {
    assert(offsets.type().id() == cudf::type_id::INT64);
    offsets_to_ranges<<<num_blocks, threads_per_block, 0, stream>>>(
      ranges_size, ranges_acc, offsets.data<int64_t>());
  }
}

}  // namespace legate::dataframe
