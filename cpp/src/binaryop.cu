#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023-2025, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <legate.h>

#include <cudf/binaryop.hpp>
#include <cudf/types.hpp>
#include <cudf/unary.hpp>

#include <legate_dataframe/binaryop.hpp>
#include <legate_dataframe/core/column.hpp>
#include <legate_dataframe/core/library.hpp>
#include <legate_dataframe/core/table.hpp>
#include <legate_dataframe/core/task_argument.hpp>
#include <legate_dataframe/core/task_context.hpp>

namespace legate::dataframe::task {

cudf::binary_operator arrow_to_cudf_binary_op(std::string op, legate::Type output_type)
{
  // Arrow binary operators taken from the below list,
  // where an equivalent cudf binary operator exists.
  // https://arrow.apache.org/docs/cpp/compute.html#element-wise-scalar-functions
  // https://docs.rapids.ai/api/libcudf/stable/group__transformation__binaryops
  std::unordered_map<std::string, cudf::binary_operator> arrow_to_cudf_ops = {
    {"add", cudf::binary_operator::ADD},
    // NOTE: if we enable true divide here, should improve polars side.
    {"divide", cudf::binary_operator::DIV},
    {"multiply", cudf::binary_operator::MUL},
    {"power", cudf::binary_operator::POW},
    {"subtract", cudf::binary_operator::SUB},
    {"bit_wise_and", cudf::binary_operator::BITWISE_AND},
    {"bit_wise_or", cudf::binary_operator::BITWISE_OR},
    {"bit_wise_xor", cudf::binary_operator::BITWISE_XOR},
    {"shift_left", cudf::binary_operator::SHIFT_LEFT},
    {"shift_right", cudf::binary_operator::SHIFT_RIGHT},
    {"logb", cudf::binary_operator::LOG_BASE},
    {"atan2", cudf::binary_operator::ATAN2},
    {"equal", cudf::binary_operator::EQUAL},
    {"greater", cudf::binary_operator::GREATER},
    {"greater_equal", cudf::binary_operator::GREATER_EQUAL},
    {"less", cudf::binary_operator::LESS},
    {"less_equal", cudf::binary_operator::LESS_EQUAL},
    {"not_equal", cudf::binary_operator::NOT_EQUAL},
    // logical operators:
    {"and", cudf::binary_operator::LOGICAL_AND},
    {"or", cudf::binary_operator::LOGICAL_OR},
    {"and_kleene", cudf::binary_operator::NULL_LOGICAL_AND},
    {"or_kleene", cudf::binary_operator::NULL_LOGICAL_OR},
  };

  // Cudf has a special case for powers with integers
  // https://github.com/rapidsai/cudf/issues/10178#issuecomment-3004143727
  if (op == "power" && output_type.to_string().find("int") != std::string::npos) {
    return cudf::binary_operator::INT_POW;
  }

  if (arrow_to_cudf_ops.find(op) != arrow_to_cudf_ops.end()) { return arrow_to_cudf_ops[op]; }
  throw std::invalid_argument("Could not find cudf binary operator matching: " + op);
  return cudf::binary_operator::INVALID_BINARY;
}

/*static*/ void BinaryOpColColTask::gpu_variant(legate::TaskContext context)
{
  TaskContext ctx{context};
  auto arrow_op  = argument::get_next_scalar<std::string>(ctx);
  const auto lhs = argument::get_next_input<PhysicalColumn>(ctx);
  const auto rhs = argument::get_next_input<PhysicalColumn>(ctx);
  auto output    = argument::get_next_output<PhysicalColumn>(ctx);
  auto op        = arrow_to_cudf_binary_op(arrow_op, output.type());

  std::unique_ptr<cudf::column> ret;
  /*
   * If one (not both) are length 1, use scalars as cudf doesn't allow
   * broadcast binary operations.
   */
  if (lhs.num_rows() == 1 && rhs.num_rows() != 1) {
    auto lhs_scalar = lhs.cudf_scalar();
    ret             = cudf::binary_operation(
      *lhs_scalar, rhs.column_view(), op, output.cudf_type(), ctx.stream(), ctx.mr());
  } else if (rhs.num_rows() == 1 && lhs.num_rows() != 1) {
    auto rhs_scalar = rhs.cudf_scalar();
    ret             = cudf::binary_operation(
      lhs.column_view(), *rhs_scalar, op, output.cudf_type(), ctx.stream(), ctx.mr());
  } else {
    ret = cudf::binary_operation(
      lhs.column_view(), rhs.column_view(), op, output.cudf_type(), ctx.stream(), ctx.mr());
  }
  if (get_prefer_eager_allocations()) {
    output.copy_into(std::move(ret));
  } else {
    output.move_into(std::move(ret));
  }
}

}  // namespace legate::dataframe::task
