#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023-2025, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/round.hpp>
#include <cudf/types.hpp>
#include <cudf/unary.hpp>
#include <legate.h>

#include <legate_dataframe/core/table.hpp>
#include <legate_dataframe/core/task_argument.hpp>
#include <legate_dataframe/core/task_context.hpp>
#include <legate_dataframe/unaryop.hpp>

namespace legate::dataframe::task {

/*static*/ void CastTask::gpu_variant(legate::TaskContext context)
{
  TaskContext ctx{context};

  const auto input                  = argument::get_next_input<PhysicalColumn>(ctx);
  auto output                       = argument::get_next_output<PhysicalColumn>(ctx);
  cudf::column_view col             = input.column_view();
  std::unique_ptr<cudf::column> ret = cudf::cast(col, output.cudf_type(), ctx.stream(), ctx.mr());
  if (get_prefer_eager_allocations()) {
    output.copy_into(std::move(ret));
  } else {
    output.move_into(std::move(ret));
  }
}

/*static*/ void RoundTask::gpu_variant(legate::TaskContext context)
{
  TaskContext ctx{context};

  const auto input      = argument::get_next_input<PhysicalColumn>(ctx);
  auto decimal_places   = argument::get_next_scalar<int32_t>(ctx);
  auto mode             = argument::get_next_scalar<std::string>(ctx);
  auto output           = argument::get_next_output<PhysicalColumn>(ctx);
  cudf::column_view col = input.column_view();
  cudf::rounding_method rounding_method;
  if (mode == "half_away_from_zero") {
    rounding_method = cudf::rounding_method::HALF_UP;
  } else if (mode == "half_to_even") {
    rounding_method = cudf::rounding_method::HALF_EVEN;
  } else {
    throw std::invalid_argument("Unsupported rounding method: " + mode);
  }
  // TODO(seberg): Need to switch to round_decimal, but it failed tests due to
  // some input types in our tests and I have not yet checked why or what to use.
  std::unique_ptr<cudf::column> ret =
    cudf::round(col, decimal_places, rounding_method, ctx.stream(), ctx.mr());
  if (get_prefer_eager_allocations()) {
    output.copy_into(std::move(ret));
  } else {
    output.move_into(std::move(ret));
  }
}

/*static*/ void UnaryOpTask::gpu_variant(legate::TaskContext context)
{
  TaskContext ctx{context};

  auto op               = argument::get_next_scalar<std::string>(ctx);
  const auto input      = argument::get_next_input<PhysicalColumn>(ctx);
  auto output           = argument::get_next_output<PhysicalColumn>(ctx);
  cudf::column_view col = input.column_view();

  // Arrow unary operators taken from the below list,
  // where an equivalent cudf unary operator exists.
  // https://arrow.apache.org/docs/cpp/compute.html#element-wise-scalar-functions
  // https://docs.rapids.ai/api/libcudf/stable/group__transformation__unaryops
  static const std::unordered_map<std::string, cudf::unary_operator> arrow_to_cudf_ops = {
    {"sin", cudf::unary_operator::SIN},       {"cos", cudf::unary_operator::COS},
    {"tan", cudf::unary_operator::TAN},       {"asin", cudf::unary_operator::ARCSIN},
    {"acos", cudf::unary_operator::ARCCOS},   {"atan", cudf::unary_operator::ARCTAN},
    {"sinh", cudf::unary_operator::SINH},     {"cosh", cudf::unary_operator::COSH},
    {"tanh", cudf::unary_operator::TANH},     {"asinh", cudf::unary_operator::ARCSINH},
    {"acosh", cudf::unary_operator::ARCCOSH}, {"atanh", cudf::unary_operator::ARCTANH},
    {"exp", cudf::unary_operator::EXP},       {"ln", cudf::unary_operator::LOG},
    {"sqrt", cudf::unary_operator::SQRT},     {"ceil", cudf::unary_operator::CEIL},
    {"floor", cudf::unary_operator::FLOOR},   {"abs", cudf::unary_operator::ABS},
    {"round", cudf::unary_operator::RINT},    {"bit_wise_not", cudf::unary_operator::BIT_INVERT},
    {"invert", cudf::unary_operator::NOT},    {"negate", cudf::unary_operator::NEGATE}};

  std::unique_ptr<cudf::column> ret;
  auto it = arrow_to_cudf_ops.find(op);
  if (it != arrow_to_cudf_ops.end()) {
    ret = cudf::unary_operation(col, it->second, ctx.stream(), ctx.mr());
  } else if (op == "is_nan") {
    ret = cudf::is_nan(col, ctx.stream(), ctx.mr());
    // As of 25.06 does not propagate nulls (historic reasons with pandas likely)
    if (col.has_nulls()) {
      auto null_mask = cudf::copy_bitmask(col, ctx.stream(), ctx.mr());
      ret->set_null_mask(std::move(null_mask), col.null_count());
    }
  } else if (op == "is_null") {
    ret = cudf::is_null(col, ctx.stream(), ctx.mr());
  } else if (op == "is_valid") {
    ret = cudf::is_valid(col, ctx.stream(), ctx.mr());
  } else {
    throw std::invalid_argument("Could not find cudf binary operator matching: " + op);
  }

  if (get_prefer_eager_allocations()) {
    output.copy_into(std::move(ret));
  } else {
    output.move_into(std::move(ret));
  }
}

}  // namespace legate::dataframe::task
